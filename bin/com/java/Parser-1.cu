#include "hip/hip_runtime.h"
package com.java;

import java_cup.runtime.Symbol;

action code
{:
	private void error(){
		parser.syntax_error(parser.symbolActual());
	}
	
	private void error(String symbol){
		parser.syntax_error(new Symbol(ParserSym.error, 
				parser.symbolActual().left, 0, symbol));
	}
:};

parser code
{:
    public Symbol symbolActual() { 
        return this.cur_token;
    }

	@Override
	public void syntax_error(Symbol s) { 
		report_error("ERR Sintaxis. Archivo: " + archivo + " Linea: " + (s.left + 1) + 
				". Texto: \"" + s.value + "\"", null);
	}

	@Override
	public void unrecovered_syntax_error(Symbol s) throws java.lang.Exception{
		System.err.println("La Cadena: \"" + s.value + "\" en la linea: " + 
				(s.left+1) + ", columna: " + s.right + " esta fuera de contexto!!!!!" );
	}

	public ParserCup(String texto, String archivo) throws Exception {
		super(new LexicoLex(new java.io.CharArrayReader(texto.toCharArray()), archivo));
		this.archivo = archivo;
	}
	
	private String archivo;	
:};

terminal String PAREN, TESIS, LLA, VES, PUNTOCOMA, COR, CHETE, PUNTO, COMILLAS, 
		MAS, MENOS, MUL, DIV, MOD, ASIGNA, COMA, DOSPUNTOS, MASIGUAL, MENOSIGUAL, 
		MULIGUAL, MODIGUAL, AND, XOR, OR, COMILLA, VCHAR, VSTRING, INT, STRING, 
		IF, ELSE, WHILE, DO, FOR, SWITCH, CASE, BREAK, CLASS, NEW, IMPORT, 
		PUBLIC, PROTECTED, PRIVATE, RETURN, PRINT, VFLOAT, VINT, ID, EXTENDS, VOID,
		APUNTADOR, CHAR, BOOLEAN, FLOAT, VBOOLEAN, COMPARA, PLUSPLUS, MINUSMINUS ;
		
terminal NEGATIVO ; 

non terminal S0, L, imports, _import, defclase, acceso, _extends, sentsc, sentc,
		atributo, tipo, atrasigna, funcion, decparams, _decparams, decparam, apunt, 
		bloque, valor, sents, sent, lclase, exp, lmatriz, _lmatriz, tupla, lfuncion,
		val, asigna, sentencias, ifelse, else, while, dowhile, for, _asigna, matriz,
		matriz_ ;

precedence left ELSE ;
precedence left OR ;
precedence left XOR ;
precedence left AND ;
precedence left COMPARA ;
precedence left MAS, MENOS ;
precedence left MUL, DIV, MOD ;
precedence left NEGATIVO ;

start with S0 ;

S0::= L ;

L::= imports defclase 
	| defclase ;

imports::= imports _import
	| _import ;

_import::= IMPORT ID PUNTOCOMA ;

defclase::= acceso CLASS ID _extends LLA sentsc VES ;

acceso::= PUBLIC
	| PRIVATE
	| PROTECTED
	| ;
	
_extends::= EXTENDS ID
	| ;

sentsc::= sentsc sentc
	| ;
	
sentc::= atributo
	| funcion ;
	
atributo::= acceso tipo ID atrasigna PUNTOCOMA 
	| acceso tipo lmatriz CHETE PUNTOCOMA ;
	
lmatriz::= lmatriz COMA valor
	| ID COR valor ;

tipo::= INT
	| STRING
	| CHAR
	| BOOLEAN
	| FLOAT
	| ID ;

atrasigna::= ASIGNA valor 
	| ;
	
funcion::= acceso tipo ID PAREN _decparams TESIS bloque 
	| acceso VOID ID PAREN _decparams TESIS bloque 
	| acceso ID PAREN _decparams TESIS bloque ;

_decparams::= decparams
	| ;

decparams::= decparams COMA decparam 
	| decparam ;
	
decparam::= tipo apunt ID ;

apunt::= APUNTADOR
	| ;

bloque::= LLA sents VES ;

sentencias::= bloque 
	| sent ;
	
sents::= sents sent 
	| ;

sent::= tipo ID atrasigna PUNTOCOMA
	| tipo matriz CHETE atrasigna PUNTOCOMA
	| matriz_ CHETE ASIGNA valor PUNTOCOMA
	| asigna PUNTOCOMA 
	| ifelse 
	| while 
	| dowhile PUNTOCOMA 
	| for ; 

matriz::= matriz COMA valor 
	| ID COR valor ;
	
matriz_::= matriz_ COMA valor
	| ID COR valor ;

asigna::= ID ASIGNA valor
	| ID MASIGUAL valor 
	| ID MENOSIGUAL valor
	| ID MULIGUAL valor 
	| ID MODIGUAL valor 
	| ID PLUSPLUS
	| PLUSPLUS ID
	| ID MINUSMINUS
	| MINUSMINUS ID ;

valor::= val
	| LLA tupla VES ;

val::= val MAS val 
	| val MENOS val 
	| val MUL val 
	| val DIV val 
	| val MOD val
	| MENOS val %prec NEGATIVO
	| exp
	| val OR val 
	| val AND val 
	| val XOR val ;
	
tupla::= tupla COMA valor 
	| valor ;

exp::= ID
	| VINT
	| VFLOAT
	| VBOOLEAN
	| COMILLA VCHAR COMILLA
	| COMILLAS VSTRING COMILLAS
	| COMILLAS COMILLAS
	| PAREN tipo TESIS valor
	| ID PLUSPLUS
	| PLUSPLUS ID
	| ID MINUSMINUS
	| MINUSMINUS ID
	| NEW lclase TESIS
	| NEW ID PAREN TESIS 
	| ID PAREN TESIS
	| lfuncion TESIS 
	| _lmatriz CHETE 
	| val COMPARA val ;
	
_lmatriz::= _lmatriz COMA valor 
	| ID COR valor ;

lclase::= lclase COMA valor
	| ID PAREN valor ;

lfuncion::= lfuncion COMA valor
	| ID PAREN valor ;

ifelse::= IF PAREN valor TESIS sentencias else ;

else::= ELSE sentencias 
	| ;
	
while::= WHILE PAREN valor TESIS sentencias ;

dowhile::= DO sentencias WHILE PAREN valor TESIS ;

for::= FOR PAREN _asigna PUNTOCOMA valor PUNTOCOMA _asigna TESIS sentencias ;

_asigna::= asigna 
	| ;

























#include "hip/hip_runtime.h"
package com.java;

import java_cup.runtime.Symbol;

action code
{:
	private void error(){
		parser.syntax_error(parser.symbolActual());
	}
	
	private void error(String symbol){
		parser.syntax_error(new Symbol(ParserSym.error, 
				parser.symbolActual().left, 0, symbol));
	}
:};

parser code
{:
    public Symbol symbolActual() { 
        return this.cur_token;
    }

	@Override
	public void syntax_error(Symbol s) { 
		report_error("ERR Sintaxis. Archivo: " + archivo + " Linea: " + (s.left + 1) + 
				". Texto: \"" + s.value + "\"", null);
	}

	@Override
	public void unrecovered_syntax_error(Symbol s) throws java.lang.Exception{
		System.err.println("La Cadena: \"" + s.value + "\" en la linea: " + 
				(s.left+1) + ", columna: " + s.right + " esta fuera de contexto!!!!!" );
	}

	public ParserCup(String texto, String archivo) throws Exception {
		super(new LexicoLex(new java.io.CharArrayReader(texto.toCharArray()), archivo));
		this.archivo = archivo;
	}
	
	private String archivo;	
:};

terminal String PAREN, TESIS, LLA, VES, PUNTOCOMA, COR, CHETE, PUNTO, COMILLAS, 
		MAS, MENOS, MUL, DIV, MOD, ASIGNA, COMA, DOSPUNTOS, MASIGUAL, MENOSIGUAL, 
		MULIGUAL, MODIGUAL, AND, XOR, OR, COMILLA, VCHAR, VSTRING, INT, STRING, 
		IF, ELSE, WHILE, DO, FOR, SWITCH, CASE, BREAK, CLASS, NEW, IMPORT, 
		PUBLIC, PROTECTED, PRIVATE, RETURN, PRINT, VFLOAT, VINT, ID, EXTENDS, VOID,
		APUNTADOR, CHAR, BOOLEAN, FLOAT, VBOOLEAN, COMPARA, PLUSPLUS, MINUSMINUS, 
		DEFAULT, NULL ;
		
terminal NEGATIVO ; 

non terminal S0, L, imports, _import, defclase, acceso, _extends, sentsc, sentc,
		atributo, tipo, atrasigna, funcion, decparams, _decparams, decparam, apunt, 
		bloque, valor, sents, sent, lclase, exp, lmatriz, _lmatriz, tupla, lfuncion,
		val, asigna, sentencias, ifelse, else, while, dowhile, for, _asigna, matriz,
		matriz_, switchcase, cases, case, default, id, func ;

precedence left ELSE ;
precedence left OR ;
precedence left XOR ;
precedence left AND ;
precedence left COMPARA ;
precedence left MAS, MENOS ;
precedence left MUL, DIV, MOD ;
precedence left NEGATIVO ;

start with S0 ;

S0::= L ;

L::= imports defclase 
	| defclase ;

imports::= imports _import
	| _import ;

_import::= IMPORT ID PUNTOCOMA ;

defclase::= acceso CLASS ID _extends LLA sentsc VES ;

acceso::= PUBLIC
	| PRIVATE
	| PROTECTED
	| ;
	
_extends::= EXTENDS ID
	| ;

sentsc::= sentsc sentc
	| ;
	
sentc::= atributo
	| funcion ;
	
atributo::= acceso tipo ID atrasigna PUNTOCOMA 
	| acceso tipo lmatriz CHETE atrasigna PUNTOCOMA ;
	
lmatriz::= lmatriz COMA valor
	| ID COR valor ;

tipo::= INT
	| STRING
	| CHAR
	| BOOLEAN
	| FLOAT
	| ID ;

atrasigna::= ASIGNA valor 
	| ;
	
funcion::= acceso tipo ID PAREN _decparams TESIS bloque 
	| acceso VOID ID PAREN _decparams TESIS bloque 
	| acceso ID PAREN _decparams TESIS bloque ;

_decparams::= decparams
	| ;

decparams::= decparams COMA decparam 
	| decparam ;
	
decparam::= tipo apunt ID ;

apunt::= APUNTADOR
	| ;

bloque::= LLA sents VES ;

sentencias::= bloque 
	| sent ;
	
sents::= sents sent 
	| ;

sent::= tipo ID atrasigna PUNTOCOMA
	| tipo matriz CHETE atrasigna PUNTOCOMA
	| asigna PUNTOCOMA 
	| id PAREN TESIS PUNTOCOMA
	| func TESIS PUNTOCOMA
	| ifelse 
	| while 
	| dowhile PUNTOCOMA 
	| for 
	| switchcase 
	| BREAK PUNTOCOMA ; 

func::= func COMA valor 
	| id PAREN valor ;

matriz::= matriz COMA valor 
	| ID COR valor ;

id::= ID
	| ID PUNTO ID ;
	
asigna::= id ASIGNA valor
	| id MASIGUAL valor 
	| id MENOSIGUAL valor
	| id MULIGUAL valor 
	| id MODIGUAL valor 
	| id PLUSPLUS
	| PLUSPLUS id
	| id MINUSMINUS
	| MINUSMINUS id
	| matriz_ CHETE ASIGNA valor
	| matriz_ CHETE MASIGUAL valor 
	| matriz_ CHETE MENOSIGUAL valor
	| matriz_ CHETE MULIGUAL valor 
	| matriz_ CHETE MODIGUAL valor 
	| matriz_ CHETE PLUSPLUS
	| PLUSPLUS matriz_ CHETE
	| matriz_ CHETE MINUSMINUS
	| MINUSMINUS matriz_ CHETE ;

matriz_::= matriz_ COMA valor
	| id COR valor ;

valor::= val
	| LLA tupla VES ;

val::= val MAS val 
	| val MENOS val 
	| val MUL val 
	| val DIV val 
	| val MOD val
	| MENOS val %prec NEGATIVO
	| exp
	| val OR val 
	| val AND val 
	| val XOR val ;
	
tupla::= tupla COMA valor 
	| valor ;

exp::= id
	| VINT
	| VFLOAT
	| VBOOLEAN
	| COMILLA VCHAR COMILLA
	| COMILLAS VSTRING COMILLAS
	| COMILLAS COMILLAS
	| PAREN tipo TESIS valor
	| id PLUSPLUS
	| PLUSPLUS id
	| id MINUSMINUS
	| MINUSMINUS id
	| NEW lclase TESIS
	| NEW ID PAREN TESIS 
	| ID PAREN TESIS
	| ID PUNTO ID PAREN TESIS 
	| lfuncion TESIS 
	| _lmatriz CHETE 
	| _lmatriz CHETE PLUSPLUS
	| PLUSPLUS _lmatriz CHETE
	| _lmatriz CHETE MINUSMINUS
	| MINUSMINUS _lmatriz CHETE 
	| val COMPARA val 
	| NULL ;
	
_lmatriz::= _lmatriz COMA valor 
	| id COR valor ;

lclase::= lclase COMA valor
	| ID PAREN valor ;

lfuncion::= lfuncion COMA valor
	| ID PAREN valor 
	| ID PUNTO ID PAREN valor ;

ifelse::= IF PAREN valor TESIS sentencias else ;

else::= ELSE sentencias 
	| ;
	
while::= WHILE PAREN valor TESIS sentencias ;

dowhile::= DO sentencias WHILE PAREN valor TESIS ;

for::= FOR PAREN _asigna PUNTOCOMA valor PUNTOCOMA _asigna TESIS sentencias ;

_asigna::= asigna 
	| ;

switchcase::= SWITCH PAREN valor TESIS LLA cases default VES ;

cases::= cases case 
	| case ;
	
case::= CASE valor DOSPUNTOS sents ;

default::= DEFAULT DOSPUNTOS sents 
	| ;























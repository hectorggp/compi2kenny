#include "hip/hip_runtime.h"
package com.formato;

import java_cup.runtime.Symbol;
import java.io.File;

action code
{:
	private int num = 0;
	
	private String tabs(){
		String ret = "";
		for(int i = 0; i < num; i++){
			ret += "\t";
		}
		return ret;
	}
	
	private void error(){
		parser.syntax_error(parser.symbolActual());
	}

	private void error(String error, int linea){
		java().error(error, parser.archivo, linea);
	}
	
	private void error(String symbol){
		parser.syntax_error(new Symbol(ParserSym.error, 
				parser.symbolActual().left, 0, symbol));
	}
	
	protected void add(String txt){
		java().add(txt);
	}
	
	protected Formador java (){ return parser.java; }

:};

parser code
{:
    public Symbol symbolActual() { 
        return this.cur_token;
    }

	@Override
	public void syntax_error(Symbol s) { 
		java.error("Símbolo '" + s.value + "' fuera de contexto", archivo, s.left + 1);
//		report_error("ERR Sintaxis. Archivo: " + archivo + " Linea: " + (s.left + 1) + 
//				". Texto: \"" + s.value + "\"", null);
	}

	@Override
	public void unrecovered_syntax_error(Symbol s) throws java.lang.Exception{
		System.err.println("La Cadena: \"" + s.value + "\" en la linea: " + 
				(s.left+1) + ", columna: " + s.right + " esta fuera de contexto!!!!!" );
	}

	public ParserCup(File file, Formador java) throws Exception {
		super(new LexicoLex(file, java));
		this.archivo = file.getName();
		this.java = java;
	}
	
	protected String archivo;	
	protected Formador java;
:};

terminal String PAREN, TESIS, LLA, VES, PUNTOCOMA, COR, CHETE, PUNTO, COMILLAS, 
		MAS, MENOS, MUL, DIV, MOD, ASIGNA, COMA, DOSPUNTOS, MASIGUAL, MENOSIGUAL, 
		MULIGUAL, MODIGUAL, AND, XOR, OR, COMILLA, VCHAR, VSTRING, INT, STRING, 
		IF, ELSE, WHILE, DO, FOR, SWITCH, CASE, BREAK, CLASS, NEW, IMPORT, 
		PUBLIC, PROTECTED, PRIVATE, RETURN, PRINT, VFLOAT, VINT, ID, EXTENDS, VOID,
		APUNTADOR, CHAR, BOOLEAN, FLOAT, VBOOLEAN, COMPARA, PLUSPLUS, MINUSMINUS, 
		DEFAULT, NULL, THIS ;
		
terminal NEGATIVO ; 

non terminal String S0, L, imports, _import, defclase, acceso, _extends, sentsc, sentc,
		atributo, tipo, atrasigna, funcion, decparams, _decparams, decparam, apunt, 
		bloque, valor, sents, sent, lclase, exp, lmatriz, _lmatriz, tupla, lfuncion,
		val, asigna, sentencias, ifelse, else, while, dowhile, for, _asigna, matriz,
		matriz_, switchcase, cases, case, default, id, func, tipoc, ves, puntocoma ;

precedence left ELSE ;
precedence left OR ;
precedence left XOR ;
precedence left AND ;
precedence left COMPARA ;
precedence left MAS, MENOS ;
precedence left MUL, DIV, MOD ;
precedence left NEGATIVO ;

start with S0 ;

S0::= L ;

L::= imports defclase 
	| defclase 
	| error {: error(); :} ;

imports::= imports _import
	| _import ;

_import::= IMPORT:i ID:j PUNTOCOMA:k {: add(i + " " + j + " " + k + "\n"); :}
	| error PUNTOCOMA {: error(); :} ;

defclase::= {: add(tabs()); :} acceso CLASS:c ID:i {: add(c + " " + i + " "); :}
		 _extends LLA {: add(" {\n"); num++; :} sentsc ves:v {: 
			num -- ;
			add("\n}");
	 		if(v != null) error("Se esperaba '}'", vleft + 1); 
		 :} ;

ves::= VES | {: RESULT = "ola q ase"; :} ;

acceso::= PUBLIC {: add("public "); :}
	| PRIVATE {: add("private "); :}
	| PROTECTED {: add("protected "); :}
	| ;
	
_extends::= EXTENDS ID:i {: add("extends " + i + " "); :}
	| ;

sentsc::= sentsc {: add(tabs()); :} sentc {: add("\n"); :}
	| ;
	
sentc::= atributo
	| funcion ;
	
atributo::= acceso tipo ID:i {: add(i); :} atrasigna puntocoma:p {: if(p != null) error("Se esperaba ';'", pleft + 1); :} 
	| acceso tipo lmatriz CHETE {: add("]"); :} atrasigna puntocoma:p {: if(p != null) error("Se esperaba ';'", pleft + 1); :}
	| error PUNTOCOMA {: error(); :};

puntocoma::= PUNTOCOMA {: add(";\n"); :} | {: add(";\n"); RESULT = "ola q ase"; :} ;
	
lmatriz::= lmatriz COMA {: add(", "); :} valor
	| ID:i COR {: add(i + " ["); :} valor ;

tipo::= INT {: add("int "); :}
	| STRING {: add("String "); :}
	| CHAR {: add("char "); :}
	| BOOLEAN {: add("boolean "); :}
	| FLOAT {: add("float "); :}
	| ID:id {: add(id + " "); :} ;

tipoc::= INT {: add("int "); :}
	| STRING {: add("String "); :}
	| CHAR {: add("char "); :}
	| BOOLEAN {: add("boolean "); :}
	| FLOAT {: add("float "); :} ;

atrasigna::= ASIGNA {: add(" = "); :} valor
	| ;
	
funcion::= acceso tipo ID:i PAREN {: add(i + " ("); :} _decparams TESIS {: add(") "); :} bloque 
	| acceso VOID ID:i PAREN {: add(i + " ("); :} _decparams TESIS {: add(") "); :} bloque 
	| acceso ID:i PAREN {: add(i + " ("); :} _decparams TESIS {: add(") "); :} bloque 
	| error {: error(); :} ;

_decparams::= decparams
	| ;

decparams::= decparams COMA {: add(", "); :} decparam 
	| decparam ;
	
decparam::= tipo apunt ID:i {: add(i); :} ;

apunt::= APUNTADOR {: add("&"); :}
	| ;

bloque::= LLA {: add("{"); num++ :} sents VES {: num--; add("\n" + tabs() + "}"); :} ;

sentencias::= bloque 
	| {: n++; add(tabs()); :} sent {: n--; :} ;
	
sents::= sents {: add(tabs()); :} sent 
	| ;

sent::= tipo ID:i {: add(i); :} atrasigna PUNTOCOMA {: add(";\n"); :}
	| tipo matriz CHETE {: add("]"); :} atrasigna PUNTOCOMA {: add(";\n"); :}
	| asigna PUNTOCOMA {: add(";\n"); :}
	| id PAREN TESIS PUNTOCOMA {: add("();\n"); :}
	| func TESIS PUNTOCOMA {: add(");\n"); :}
	| ifelse 
	| while 
	| dowhile PUNTOCOMA {: add(";\n"); :}
	| for 
	| switchcase 
	| BREAK PUNTOCOMA {: add("break;\n"); :}
	| RETURN {: add("return "); :} valor PUNTOCOMA {: add(";\n"); :}
	| PRINT PAREN {: add("print ("); :} val TESIS PUNTOCOMA {: add(");\n"); :}
	| error PUNTOCOMA {: error(); :} ; 

func::= func COMA {: add(", "); :} valor 
	| id PAREN {: add("("); :} valor ;

matriz::= matriz COMA {: add(", "); :} valor 
	| ID:i COR {: add(i + " [" :} valor ;

id::= ID:i {: add(i); :}
	| ID:id1 PUNTO ID:id2 {: add(id1 + "." + id2); :} 
	| THIS PUNTO ID:i {: add("this." + i); :} ;
	
asigna::= id ASIGNA {: add(" = "); :} valor
	| id MASIGUAL {: add(" += "); :} valor 
	| id MENOSIGUAL {: add(" -= "); :} valor
	| id MULIGUAL {: add(" *= "); :} valor 
	| id MODIGUAL {: add(" %= "); :} valor 
	| id PLUSPLUS {: add(" ++"); :}
	| PLUSPLUS {: add("++ "); :} id
	| id MINUSMINUS {: add(" --"); :}
	| MINUSMINUS {: add("-- "); :} id
	| matriz_ CHETE ASIGNA {: add("] = "); :} valor
	| matriz_ CHETE MASIGUAL {: add("] += "); :} valor 
	| matriz_ CHETE MENOSIGUAL {: add("] -= "); :} valor
	| matriz_ CHETE MULIGUAL {: add("] *= "); :} valor 
	| matriz_ CHETE MODIGUAL {: add("] %= "); :} valor 
	| matriz_ CHETE PLUSPLUS {: add("] ++"); :}
	| PLUSPLUS {: add("++ "); :}  matriz_ CHETE {: add("]"); :}
	| matriz_ CHETE MINUSMINUS {: add("] --"); :}
	| MINUSMINUS {: add("-- "); :} matriz_ CHETE {: add("]"); :} ;

matriz_::= matriz_ COMA {: add(", "); :} valor
	| id:i COR {: add(i + " ["); :} valor ;

valor::= val
	| LLA {: add("{"); :} tupla VES {: add("}"); :} 
	| error {: error(); :} ;

val::= val MAS {: add(" + "); :} val 
	| val MENOS {: add(" - "); :} val 
	| val MUL {: add(" * "); :} val 
	| val DIV {: add(" / "); :} val 
	| val MOD {: add(" % "); :} val 
	| MENOS {: add(" - "); :} val %prec NEGATIVO 
	| PAREN {: add("("); :} val TESIS {: add(")"); :}
	| exp 
	| val COMPARA:c {: add(c + " "); :} val 
	| val OR {: add(" || "); :} val 
	| val AND {: add(" && "); :} val 
	| val XOR {: add(" ^ "); :} val ;
	
tupla::= tupla COMA {: add(", "); :} valor 
	| valor ;

exp::= id
	| VINT:v {: add(v); :}
	| VFLOAT:v {: add(v); :}
	| VBOOLEAN:v {: add(v); :}
	| COMILLA VCHAR:v COMILLA {: add("'" + v + "'"); :}
	| COMILLAS VSTRING:v COMILLAS {: add("\"" + v + "\""); :}
	| COMILLAS COMILLAS {: add("\"\""); :}
	| PAREN {: add("("); :} tipoc TESIS {: add(") "); :} valor 
	| id PLUSPLUS {: add(" ++"); :}
	| PLUSPLUS {: add("++ "); :} id
	| id MINUSMINUS {: add(" --"); :}
	| MINUSMINUS {: add("-- "); :} id
	| NEW {: add("new "); :} lclase TESIS {: add(")"); :}
	| NEW ID:i PAREN TESIS {: add("new " + i + "()"); :}
	| ID:i PAREN TESIS {: add(i + "()"); :}
	| ID:i1 PUNTO ID:i2 PAREN TESIS {: add(i1 + "." + i2 + "()"); :}
	| lfuncion TESIS {: add(")"); :}
	| _lmatriz CHETE 
	| _lmatriz CHETE PLUSPLUS
	| PLUSPLUS _lmatriz CHETE
	| _lmatriz CHETE MINUSMINUS
	| MINUSMINUS _lmatriz CHETE 
	| NULL ;
	
_lmatriz::= _lmatriz COMA valor 
	| id COR valor ;

lclase::= lclase COMA valor
	| ID PAREN valor ;

lfuncion::= lfuncion COMA valor
	| ID PAREN valor 
	| ID PUNTO ID PAREN valor ;

ifelse::= IF PAREN valor TESIS sentencias else ;

else::= ELSE sentencias 
	| ;
	
while::= WHILE PAREN valor TESIS sentencias ;

dowhile::= DO sentencias WHILE PAREN valor TESIS ;

for::= FOR PAREN _asigna PUNTOCOMA valor PUNTOCOMA _asigna TESIS sentencias ;

_asigna::= asigna 
	| ;

switchcase::= SWITCH PAREN valor TESIS LLA cases default VES ;

cases::= cases case 
	| case ;
	
case::= CASE valor DOSPUNTOS sents ;

default::= DEFAULT DOSPUNTOS sents 
	| ;























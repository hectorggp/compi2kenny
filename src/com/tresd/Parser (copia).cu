#include "hip/hip_runtime.h"
package com.tresd;

import java_cup.runtime.Symbol;
import java.io.File;

action code
{:
	private void error(){
		parser.syntax_error(parser.symbolActual());
	}

	private void error(String error, int linea){
		clase().error(error, parser.archivo, linea);
	}
	
	private void error(String symbol){
		parser.syntax_error(new Symbol(ParserSym.error, 
				parser.symbolActual().left, 0, symbol));
	}
	
	private Clase clase (){ return parser.clase; }
	private String nclase(){ return parser.nclase; }
	private int pasada(){ return parser.pasada; }
	private Funcion funcion(){ return parser.funcion; }
	private void funcion(Funcion f) { parser.funcion = f; }
	private void add(String txt) { clase().add(txt); }
	private void addt(String txt) { add("\t" + txt + "\n"); }
	private String genTemp(){ return Clase.genTemp(); }
	private String genEtiq(){ return Clase.genEtiq(); }
	private void resetcTemp() { Clase.resetcTemp(); }
	private String apheap(){ return Clase.apHeap; }
	private String apstack(){ return Clase.apStack; }
	private String heap(){ return Clase.Heap; }
	private String stack(){ return Clase.Stack; }
	private String escape(){ return "-2.71"; }

:};

parser code
{:
    public Symbol symbolActual() { 
        return this.cur_token;
    }

	@Override
	public void syntax_error(Symbol s) { 
		clase.error("Símbolo '" + s.value + "' fuera de contexto", archivo, s.left + 1);
//		report_error("ERR Sintaxis. Archivo: " + archivo + " Linea: " + (s.left + 1) + 
//				". Texto: \"" + s.value + "\"", null);
	}

	@Override
	public void unrecovered_syntax_error(Symbol s) throws java.lang.Exception{
		System.err.println("La Cadena: \"" + s.value + "\" en la linea: " + 
				(s.left+1) + ", columna: " + s.right + " esta fuera de contexto!!!!!" );
	}

	public ParserCup(File file, Clase clase, int pasada) throws Exception {
		super(new LexicoLex(file, clase));
		this.archivo = file.getName();
		this.clase = clase;
		this.nclase = this.archivo.split("\\.")[0];
		this.pasada = pasada;
	}
	
	protected String archivo;	
	protected Clase clase;
	protected String nclase;
	protected int pasada;
	protected Funcion funcion;
:};

terminal String PAREN, TESIS, LLA, VES, PUNTOCOMA, COR, CHETE, PUNTO, COMILLAS, 
		MAS, MENOS, MUL, DIV, MOD, ASIGNA, COMA, DOSPUNTOS, MASIGUAL, MENOSIGUAL, 
		MULIGUAL, MODIGUAL, AND, XOR, OR, COMILLA, VCHAR, VSTRING, INT, STRING, 
		IF, ELSE, WHILE, DO, FOR, SWITCH, CASE, BREAK, CLASS, NEW, IMPORT, 
		PUBLIC, PROTECTED, PRIVATE, RETURN, PRINT, VFLOAT, VINT, ID, EXTENDS, VOID,
		APUNTADOR, CHAR, BOOLEAN, FLOAT, VBOOLEAN, COMPARA, PLUSPLUS, MINUSMINUS, 
		DEFAULT, NULL, THIS ;
		
terminal NEGATIVO ; 

non terminal S0, L, imports, _import, defclase, _extends, sentsc, sentc,
		atributo, atrasigna, funcion, declasig,
		bloque, sents, sent, lclase, _lmatriz, tupla, lfuncion, puntocoma,
		asigna, sentencias, ifelse, else, while, dowhile, for, _asigna, matriz,
		matriz_, switchcase, cases, case, default, id, idvalor, func, tipoc, ves ; 
non terminal Integer valorn ;
non terminal String lmatriz, decparam, decparams, apunt, _decparams, tipo ;
non terminal Character acceso ;
non terminal Valor val, valor, exp, decl ;

precedence left ELSE ;
precedence left OR ;
precedence left XOR ;
precedence left AND ;
precedence left COMPARA ;
precedence left MAS, MENOS ;
precedence left MUL, DIV, MOD ;
precedence left NEGATIVO ;

start with S0 ;

S0::= L ;

L::= imports defclase 
	| defclase 
	| error {: error(); :} ;

imports::= imports _import
	| _import ;

_import::= IMPORT ID:id {:
		if(pasada() == 1) 
			clase().addClase(id, idleft + 1); 
	:} PUNTOCOMA 
	| error PUNTOCOMA {: error(); :} ;

defclase::= CLASS ID:id {: 
		if(pasada() == 1)
			clase().revisaNombre(id, idleft + 1); 
	:} _extends LLA sentsc ves:v {: if(v != null) error("Se esperaba '}'", vleft + 1); :} ;

ves::= VES | {: RESULT = "ola q ase"; :} ;

acceso::= PUBLIC {: if(pasada() == 1 || pasada() == 2) RESULT = Variable.vpublic; :}
	| PRIVATE {: if(pasada() == 1 || pasada() == 2) RESULT = Variable.vprivate; :}
	| PROTECTED {: if(pasada() == 1 || pasada() == 2) RESULT = Variable.vprotected; :}
	| {: if(pasada() == 1 || pasada() == 2) RESULT = Variable.vpublic; :} ;
	
_extends::= EXTENDS ID:id {: 
		if(pasada() == 1)
			clase().claseExtiende(id, idleft + 1);
	:}
	| ;

sentsc::= sentsc sentc
	| ;
	
sentc::= atributo
	| funcion ;
	
atributo::= acceso:a tipo:t ID:id {:
		if(pasada() == 1)
			clase().agregaAtributo(a, t, id, idleft + 1);
	:} atrasigna puntocoma:p {: if(p != null) error("Se esperaba ';'", pleft + 1); :} 
	| acceso:a tipo:t lmatriz:l CHETE atrasigna {:
		if(pasada() == 1 && l != null) 
			clase().agregaAtributo(a, t, l, lleft + 1);
	:} puntocoma:p {: if(p != null) error("Se esperaba ';'", pleft + 1); :}
	| error PUNTOCOMA {: error(); :};

puntocoma::= PUNTOCOMA | {: RESULT = "ola q ase"; :} ;
	
lmatriz::= lmatriz:l COMA valorn:v {: 
		if(pasada() == 1) {
			if(v > 0){
				if (l != null)
					RESULT = l + ":" + v; 
			} else 
				error("Dimensión no válida. Se esperaba un valor positivo diferente de cero", vleft + 1);
		}
	:}
	| ID:id COR valorn:v {: 
		if(pasada() == 1) {
			if(v > 0)
				RESULT = id + "-" + v; 
			else 
				error("Dimensión no válida. Se esperaba un valor positivo diferente de cero", vleft + 1);
		}
	:} ;

valorn::= VINT:i {: if(pasada() == 1) RESULT = Integer.valueOf(i); :} 
	| valorn:v1 MAS valorn:v2 {: if(pasada() == 1) RESULT = v1 + v2; :}
	| valorn:v1 MENOS valorn:v2 {: if(pasada() == 1) RESULT = v1 - v2; :}
	| valorn:v1 MUL valorn:v2 {: if(pasada() == 1) RESULT = v1 * v2; :}
	| valorn:v1 DIV valorn:v2 {: if(pasada() == 1) RESULT = v1 / v2; :}
	| valorn:v1 MOD valorn:v2 {: if(pasada() == 1) RESULT = v1 % v2; :}
	| PAREN valorn:v TESIS {: if(pasada() == 1) RESULT = v; :}
	| MENOS valorn:v {: if(pasada() == 1) RESULT = - v; :} %prec NEGATIVO 
	| error {: RESULT = 0; :} ;

tipo::= INT {: if(pasada() == 1 || pasada() == 2) RESULT = Variable.tint; :}
	| STRING {: if(pasada() == 1 || pasada() == 2) RESULT = Variable.tString; :}
	| CHAR {: if(pasada() == 1 || pasada() == 2) RESULT = Variable.tchar; :}
	| BOOLEAN {: if(pasada() == 1 || pasada() == 2) RESULT = Variable.tboolean; :}
	| FLOAT {: if(pasada() == 1 || pasada() == 2) RESULT = Variable.tfloat; :}
	| ID:id {: if(pasada() == 1 || pasada() == 2) RESULT = id; :} ;

tipoc::= INT {: if(pasada() == 1) RESULT = Variable.tint; :}
	| STRING {: if(pasada() == 1) RESULT = Variable.tString; :}
	| CHAR {: if(pasada() == 1) RESULT = Variable.tchar; :}
	| BOOLEAN {: if(pasada() == 1) RESULT = Variable.tboolean; :}
	| FLOAT {: if(pasada() == 1) RESULT = Variable.tfloat; :} ;

atrasigna::= ASIGNA valor 
	| ;
	
funcion::= acceso:a tipo:t ID:i PAREN {:
		if(pasada() == 2) {
			funcion(new Funcion(clase()));
			add("\n" + t + "_" + i);
		}
	:} _decparams:p {:
		if(pasada() == 1)
			clase().addFuncion(a, t, i, p, ileft + 1);
	:} TESIS bloque {: if(pasada() == 2) { add("}\n"); resetcTemp(); } :}
	| acceso:a VOID:t ID:i PAREN {:
		if(pasada() == 2) {
			funcion(new Funcion(clase()));
			add("\n" + t + "_" + i);
		}
	:} _decparams:p {:
		if(pasada() == 1)
			clase().addFuncion(a, t, i, p, ileft + 1);
	:} TESIS bloque {: if(pasada() == 2) { add("}\n"); resetcTemp(); } :} 
	| acceso:a ID:i PAREN {:
		if(pasada() == 2) {
			funcion(new Funcion(clase()));
			add("\n" + "void_" + i);
		}
	:} _decparams:p {:
		if(pasada() == 1)
			clase().addFuncion(a, "%constructor%", i, p, ileft + 1);
	:} TESIS bloque {: if(pasada() == 2) { add("}\n"); resetcTemp(); } :} 
	| error {: error(); :} ;

_decparams::= decparams:d {: 
		if (pasada() == 1) 
			RESULT = d; 
		else if (pasada() == 2) 
			add("() {\n");
	:}
	| {: 
		if(pasada() == 1) 
			RESULT = ""; 
		else if (pasada() == 2)
			add("() {\n");
	:} ;

decparams::= decparams:ds COMA decparam:d {: RESULT = ds + ":" + d; :}
	| decparam:d {: RESULT = d; :} ;
	
decparam::= tipo:t apunt:a ID:id {: 
		if(pasada() == 1) 
			RESULT = a + t; 
		else if(pasada() == 2) {
			funcion().addVariable(t, id, idleft + 1);
			add("_" + t);
		}
	:} ;

apunt::= APUNTADOR {: if(pasada() == 1) RESULT = "&"; :}
	| {: if(pasada() == 1) RESULT = " "; :} ;

bloque::= LLA sents VES ;

sentencias::= sent | bloque ;
	
sents::= sents sent 
	| ;

declasig::= decl:id ASIGNA valor:v {: 
		if(pasada() == 2) {
			if(id != null && v != null) {
				if(id.tipo.equals(v.tipo)) {
					if(id.tipo.equals(Variable.tString)){
						if(v.primitivo){ // se copia cada posición del string
							int i = 0;
							String tem = id.temp;
							String t = genTemp();
							for(char c : v.texto.toCharArray()) {
								addt(t + " = " + tem + " + " + i);
								addt(stack() + "[" + t + "] = " + (int) c);
								if(i == 256) 
									break;
								i ++;
							}
							addt(t + " = " + tem + " + " + i);
							addt(stack() + "[" + t + "] = " + escape());
						}
					}
				} else
					error("Los operandos son tipos diferentes", idleft + 1);
			}
		}
	:}
	| decl ;

decl::= tipo:t ID:id {: // retorna texto de la forma stack[tn] con su tipo
		if(pasada() == 2) {
			if(funcion().addVariable(t, id, idleft + 1)) {
				int pos = funcion().getPosicionStackVariable(id);
				if(pos >= 0) {
					String t1 = genTemp();
					addt(t1 + " = " + apstack() + " + " + pos);
					RESULT = new Valor(stack() + "[" + t1 + "]", t);
				}
			}
		}
	:}
	| matriz CHETE ;

sent::= declasig PUNTOCOMA
	| asigna PUNTOCOMA 
	| id PAREN TESIS PUNTOCOMA
	| func TESIS PUNTOCOMA
	| ifelse 
	| while 
	| dowhile PUNTOCOMA 
	| for 
	| switchcase 
	| BREAK PUNTOCOMA 
	| RETURN valor PUNTOCOMA
	| PRINT PAREN val TESIS PUNTOCOMA 
	| error PUNTOCOMA {: error(); :} ; 

func::= func COMA valor 
	| id PAREN valor ;

matriz::= matriz COMA valor 
	| tipo ID COR valor ;

id::= ID // retorna un string tipo 'stack[tn]'
	| ID PUNTO ID 
	| THIS PUNTO ID ;
	
idvalor::= ID:id {: // retorna un string tipo tn
		if(pasada() == 2) {
			int pos = funcion().getPosicionStackVariable(id);
			if(pos >= 0){
				String t = genTemp();
			} else {
				pos = clase().getPosicionHeapAtributo(id, idleft + 1);
				if(pos >= 0){
				
				}
			}
		}
	:} 
	| THIS PUNTO ID PUNTO ID {::}
	| ID:ins PUNTO ID:id {::}
	| THIS PUNTO ID{::} ;
	
asigna::= id ASIGNA valor
	| id MASIGUAL valor 
	| id MENOSIGUAL valor
	| id MULIGUAL valor 
	| id MODIGUAL valor 
	| id PLUSPLUS
	| PLUSPLUS id
	| id MINUSMINUS
	| MINUSMINUS id
	| matriz_ CHETE ASIGNA valor
	| matriz_ CHETE MASIGUAL valor 
	| matriz_ CHETE MENOSIGUAL valor
	| matriz_ CHETE MULIGUAL valor 
	| matriz_ CHETE MODIGUAL valor 
	| matriz_ CHETE PLUSPLUS
	| PLUSPLUS matriz_ CHETE
	| matriz_ CHETE MINUSMINUS
	| MINUSMINUS matriz_ CHETE ;

matriz_::= matriz_ COMA valor // retorna un string tipo 'stack[tn]'
	| ID COR valor 
	| ID PUNTO ID COR valor 
	| THIS PUNTO ID COR valor ;

valor::= val:v {: if(pasada() == 2) RESULT = v; :} // retorna un Valor de la forma 'tn' con su tipo. Pueden venir valores primitivos
	| LLA tupla VES 
	| error {: error(); :} ;

val::= val MAS val 
	| val MENOS val 
	| val MUL val 
	| val DIV val 
	| val MOD val 
	| MENOS val %prec NEGATIVO 
	| PAREN val TESIS 
	| exp:e {: if(pasada() == 2) RESULT = e; :}
	| val COMPARA val 
	| val OR val 
	| val AND val 
	| val XOR val ;
	
tupla::= tupla COMA valor 
	| valor ;

exp::= idvalor
	| VINT
	| VFLOAT
	| VBOOLEAN
	| COMILLA VCHAR COMILLA
	| COMILLAS VSTRING:v {: if(pasada() == 2) RESULT = new Valor(v, Variable.tString); :} COMILLAS
	| COMILLAS COMILLAS
	| PAREN tipoc TESIS valor
	| idvalor PLUSPLUS
	| PLUSPLUS idvalor
	| idvalor MINUSMINUS
	| MINUSMINUS idvalor
	| NEW lclase TESIS
	| NEW ID PAREN TESIS 
	| ID PAREN TESIS
	| ID PUNTO ID PAREN TESIS 
	| lfuncion TESIS 
	| _lmatriz CHETE 
	| _lmatriz CHETE PLUSPLUS
	| PLUSPLUS _lmatriz CHETE
	| _lmatriz CHETE MINUSMINUS
	| MINUSMINUS _lmatriz CHETE 
	| NULL ;
	
_lmatriz::= _lmatriz COMA valor 
	| idvalor COR valor ;

lclase::= lclase COMA valor
	| ID PAREN valor ;

lfuncion::= lfuncion COMA valor
	| ID PAREN valor 
	| ID PUNTO ID PAREN valor ;

ifelse::= IF PAREN valor TESIS sentencias else ;

else::= ELSE sentencias 
	| ;
	
while::= WHILE PAREN valor TESIS sentencias ;

dowhile::= DO sentencias WHILE PAREN valor TESIS ;

for::= FOR PAREN _asigna PUNTOCOMA valor PUNTOCOMA _asigna TESIS sentencias ;

_asigna::= asigna 
	| ;

switchcase::= SWITCH PAREN valor TESIS LLA cases default VES ;

cases::= cases case 
	| case ;
	
case::= CASE valor DOSPUNTOS sents ;

default::= DEFAULT DOSPUNTOS sents 
	| ;


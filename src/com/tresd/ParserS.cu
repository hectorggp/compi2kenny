#include "hip/hip_runtime.h"
package com.tresd;

import java_cup.runtime.Symbol;
import java.io.File;
import java.util.ArrayList;

action code
{:
	private void error(){
		parser.syntax_error(parser.symbolActual());
	}

	private void error(String error, int linea){
		clase().error(error, parser.archivo, linea);
	}
	
	private void error(String symbol){
		parser.syntax_error(new Symbol(ParserSym.error, 
				parser.symbolActual().left, 0, symbol));
	}
	
	private Variable [] cases = new Variable [50];
	private int icases = -1;
	private String [] display = new String [50];
	private int idisplay = -1;
	private String [] pila = new String [50];		// para push
	private int ipila = -1;							// y pop
	private String tempPorGuardar = "";
	private String etSalidaMetodo;
	
	private void addtif(String val, String bool, String et){
		addt("if (" + val + " == " + bool + ") goto " + et);
	}
	
	private void push(String val){
		pila[++ ipila] = val;
	}
	
	private String pop(){
		return pila[ipila --];
	}
	
	private Clase clase(){ return parser.clase; }
	private String nclase(){ return parser.nclase; }
	private int pasada(){ return parser.pasada; }
	private Funcion funcion(){ return parser.funcion; }
	private void funcion(Funcion f) { parser.funcion = f; }
	private void add(String txt) { clase().add(txt); }
	private void addt(String txt) { add("\t" + txt + ";\n"); }
	private String genTemp(boolean guardar){
		String ret = Clase.genTemp();
		if(guardar)
			tempPorGuardar += tempPorGuardar.equals("") ? ret : ":" + ret;
		return ret;
	}
	private String genTemp(){ return genTemp(true); }
	private void resetTempPorGuardar(){ tempPorGuardar = ""; }
	private String genEtiq(){ return Clase.genEtiq(); }
	private void resetcTemp() { Clase.resetcTemp(); }
	private String apheap(){ return Clase.apHeap; }
	private String apstack(){ return Clase.apStack; }
	private String heap(){ return Clase.Heap; }
	private String stack(){ return Clase.Stack; }
	private String escape(){ return String.valueOf(Variable.escapeString); }

	/**
	 * Realiza el protocolo de llamada a función. Retorna null 
	 * si es un procedimiento sin retorno. Si no, genera variable  
	 * @param clase clase de la instancia a la que pertenece el método
	 * @param metodo método al que se llama
	 * @param pars lista de instancias de Variable puestos como parámetros
	 * @param posInstancia temporal que posee la posición de la instancia
	 * @param linea linea por posible error
	 * @return
	 */
	private Variable llamarFuncion(String clase, String metodo, 
			ArrayList<Variable> pars, String posInstancia, int linea){
		Variable ret = null;
		/**
		 * 1) verificar que metodo existe y que es accesible
		 * 2) guardar temps en stack
		 * 3) setear parametros por valor o referencia
		 * 4) actualizar punteros de stack y heap
		 * 5) llamar procedimiento
		 * 6) tomar valor de retorno
		 * 7) restablecer temporales
		 * 8) retornar valor 
		 */
		String def = clase().definicionDeClaseSiPuede(clase, metodo, pars, linea);
		if (def != null) { // ya se trató el error
							// def de la forma tipoRetorno-nombre[-[&| ]t1:[&| ]t2: ... :[&| ]tn]
			String [] vdef = def.split("-");
			int ind_ = vdef[0].indexOf('_');
			String clasep = vdef[0].substring(0, ind_);
			String retorno = vdef[0].substring(ind_ + 1);
			String [] vpars = vdef.length == 3 ? vdef[2].split(":") : null;
			String temp = genTemp(false);
			int tamAmbito = funcion().getTamaño();
			String [] vtempPorGuardar = tempPorGuardar.split(":");
			
			addt("// guardar temps en stack");
			// 2) guardar temps en stack
			if(!tempPorGuardar.equals("")){
				for(int i = 0; i < vtempPorGuardar.length; i++){
					addt(temp + " = " + apstack() + " + " + tamAmbito ++);
					addt(stack() + "[(int) " + temp + "] = " + vtempPorGuardar[i]);
				}
			}
			
			addt("// setear parametros por valor o referencia");
			// 3) setear parametros por valor o referencia
			int posStack = tamAmbito;
			if(!retorno.equals(Funcion.rconstructor) && !retorno.equals(Funcion.rvoid))
				posStack ++;
			String defin = "";
			for(int i = 0; i < pars.size(); i++){
				addt(temp + " = " + apstack() + " + " + ++ posStack);
				Variable par = pars.get(i);
				defin += "_" + par.tipo;
				if(vpars[i].charAt(0) == ' ' && Variable.esPrimitiva(par.tipo)){
					if(par.lugar == null){// valores float simples 
						addt(stack() + "[(int) " + temp + "] = " + par.temp);
					} else { // su 'temp' está apuntando a 'lugar'
						if(!par.referencia){ // sólo primitivos
							addt(par.temp + " = " + par.lugar + "[(int) " + par.temp + "]");
							addt(stack() + "[(int) " + temp + "] = " + par.temp);
						} else { // solo primitivos en stack
							addt(par.temp + " = " + stack() + "[(int) " + par.temp + "]");
							addt(par.temp + " = " + stack() + "[(int) " + par.temp + "]");
							addt(stack() + "[(int) " + temp + "] = " + par.temp);
						}
					}
				} else { // espera referencia de primitivo en stack o es instancia de clase
					if(par.lugar == null){
						error("Se espera variable", linea);
					} else {
						if(Variable.esPrimitiva(par.tipo)){
							if(!par.referencia) {
								addt(stack() + "[(int) " + temp + "] = " + par.temp);
							} else {
								addt(par.temp + " = " + stack() + "[(int) " + par.temp + "]");
								addt(stack() + "[(int) " + temp + "] = " + par.temp);
							}
						} else { // es instancia de clase o String
							if(par.lugar.equals(stack()))
								add(par.temp + " = " + par.lugar + "[(int) " + par.temp + "]");
							addt(stack() + "[(int) " + temp + "] = " + par.temp);
						}
					}
				} 
			}
			
			addt("// actualizar punteros en stack y heap");
			// 4) actualizar punteros de stack y heap
			addt(temp + " = " + apstack() + " + " + tamAmbito);
			addt(stack() + "[(int) " + temp + "] = " + posInstancia);
			addt(apstack() + " = " + apstack() + " + " + tamAmbito);
			
			addt("// llamar al procedimiento");
			// 5) llamar al procedimiento
			metodo = clasep + "_" + (retorno.equals(Funcion.rconstructor) ? 
					"void" : retorno) + "_" + metodo + defin;
			addt(metodo + "()");
			addt("// restablecer valor de apstack");
			// 5.1) restablecer valor de apstack
			addt(apstack() + " = " + apstack() + " - " + tamAmbito);
			
			addt("// tomar valor retorno");
			// 6) tomar valor retorno
			if(!retorno.equals(Funcion.rconstructor) && !retorno.equals(Funcion.rvoid)){
				String tret = genTemp();
				addt(tret + " = " + apstack() + " + " + (tamAmbito + 1));
				ret = new Variable(tret, Variable.esPrimitiva(retorno) ?
						stack() : heap(), retorno, !Variable.esPrimitiva(retorno), false);
			}
			
			addt("// restablecer temporales");
			// 7) restablecer temporales
			if(!tempPorGuardar.equals("")){
				for(int i = vtempPorGuardar.length - 1; i >= 0; i --){
					addt(temp + " = " + apstack() + " + " + -- tamAmbito);
					addt(vtempPorGuardar[i] + " = " + stack() + "[(int) " + temp + "]");
				}
			}
		}
		// 8) retornar valor
		return ret;
	}

	/**
	 * retorna temporal con el valor de la posición en heap de la nueva instancia
	 * Actualiza el aputador al heap
	 * @param sclase clase a instanciar
	 * @param nulo si la nueva clase está inicializada como null
	 * @param linea liea por posible error
	 * @return temporal generado o null si no existe clase
	 */
	private String tnuevaInstancia(String sclase, boolean nulo, int linea){
		int tamClase = sclase.equals(Variable.tString) ? 
				Variable.tamString : clase().tamClase(sclase);
		if(tamClase > 0){
			String temp = genTemp();
			addt(temp + " = " + apheap());
			if(!nulo)
				addt(heap() + "[(int) " + temp + "] = " + temp);
			else
				addt(heap() + "[(int) " + temp + "] = -1");
			addt(apheap() + " = " + apheap() + " + " + tamClase);
			return temp;
		}
		error("No se encuentra clase '" + sclase + "'", linea);
		return null;
	}
	
	/**
	 * retorna temporal con el valor de la posición en heap de la nueva instancia
	 * Actualiza el aputador al heap
	 * @param sclase clase a instanciar
	 * @param linea liea por posible error
	 * @return temporal generado o null si no existe clase
	 */
	private String tnuevaInstancia(String sclase, int linea){
		return tnuevaInstancia(sclase, false, linea);
	}
	
	/**
	 * Retorna un temporal con el valor de la posición en heap donde empieza 
	 * el string
	 * @param valor la cadena String
	 * @param linea la linea por posible error
	 * @return el temporal 
	 */
	private String tnuevaInstanciaString(String valor, int linea){
		String tstring = tnuevaInstancia(Variable.tString, linea);
		String temp = genTemp(false);
		int i = 0;
		for(char c : valor.toCharArray()){
			addt(temp + " = " + tstring + " + " + ++ i);
			// addt(heap() + "[(int) " + temp + "] = " + (int) c);
			addt(heap() + "[(int) " + temp + "] = " + (int)c);
		}
		addt(temp + " = " + tstring + " + " + ++ i);
		addt(heap() + "[(int) " + temp + "] = " + escape());
		return tstring;
	}

	/**
	 * Retorna Variable, el temporal tiene el valor correspondiente al resultado de la operación
	 * de tipo ++ o --
	 * Retorna null si la Variable es inválida
	 * @param antes Si el operador está antes de la variable
	 * @param op el operador, " + " o " - "
	 * @param v la Variable debe ser variable
	 * @param linea linea por posible error
	 * @return Variable cuyo temporal tiene el resultado de la operación
	 */
	private Variable cvariable(boolean antes, String op, Variable v, int linea){
		Variable ret = null;
		String temp1 = genTemp(!antes);
		String temp2 = genTemp(antes);
		if(!v.referencia){ // es un valor en 'lugar' apuntado por 'temp'
			addt(temp1 + " = " + v.lugar + "[(int) " + v.temp + "]");
			addt(temp2 + " = " + temp1 + op + 1);
			addt(v.lugar + "[(int) " + v.temp + "] = " + temp2);
			ret = new Variable(antes ? temp2 : temp1, null, v.tipo, false, false);
		} else { // es una referencia en stack apuntada por 'temp'
			if(v.lugar.equals(stack())){
				addt(temp1 + " = " + stack() + "[(int) " + v.temp + "]");
				addt(temp1 + " = " + stack() + "[(int) " + temp1 + "]");
				addt(temp2 + " = " + temp1 + op + 1);
				addt(stack() + "[(int) " + v.temp + "] = " + temp2);
				ret = new Variable(antes ? temp2 : temp1, null, v.tipo, false, false);
			} else
				error("Se esperaba variable numérica de ámbito", linea + 1);
		}
		return ret;
	}
	
	/**
	* Busca en los ámtibos actuales la variable etiquetada con 'id'
	* Si no se encuentra, devuelve null
	* @param id Etiqueta de la variable a buscar
	* @return Variable con los datos de la variable buscada o null si no se encuentra
	*/
	private Variable buscaEnAmbito(String id){
		Variable ret = null;
		Variable var = funcion().getPosicionVariable(id);
		if(var != null){
			String [] vatr = var.definición.split("-");
			String def = vatr.length > 1 ? "-" + vatr[1] : "";
			String temp = genTemp();
			addt(temp + " = " + apstack() + " + " + var.getPosición());
			ret = new Variable(temp + def, stack(), var.tipo, var.referencia, true);
		}
		return ret;
	}
	
	/**
	 * Busca el atributo de la instancia 'clasePoseedora' a partir de una instancia 
	 * 'claseDeOrigen'. Valida que se encuentre y visibilidad. Si es inválido, devuelve null
	 * @param id etiqueta del atributo 
	 * @param clasePoseedora clase que posee el atributo
	 * @param temp temporal que posee el valor de la posición del heap donde está la instancia 'clasePoseedora'
	 * @param linea linea por posible error
	 * @return Variable si se encuentra, o null si hay error
	 */
	private Variable buscaEnInstancia(String id, String clasePoseedora, String temp, int linea){
		Variable ret = null;
		Variable atr = clase().buscarEnClase(id, clasePoseedora, linea);
		if(atr != null){
			String [] vatr = atr.definición.split("-");
			String def = vatr.length > 1 ? "-" + vatr[1] : "";
			addt(temp + " = " + temp + " + " + atr.getPosición());
			ret = new Variable(temp + def, heap(), atr.tipo, !Variable.esPrimitiva(atr.tipo), true);
		}
		return ret;
	}

	/**
	 * Asigna al lugar apuntado por var.temp, en el lugar var.lugar, el valor o apuntador del lugar en
	 * val.lugar apuntado por  val.temp
	 * @param var Variable donde será asignado el nuevo valor
	 * @param val Variable que posee el valor a asignar
	 */
	private void asigna(Variable var, Variable val, String op, int linea){
		if(var.esPrimitiva()){
			if(val.temp != null && var.tipo.equals(val.tipo)){
				if(val.lugar == null){
				} else if(!val.referencia){
					addt(val.temp + " = " + val.lugar + "[(int) " + val.temp + "]");
				} else {
					addt(val.temp + " = " + stack() + "[(int) " + val.temp + "]");
					addt(val.temp + " = " + stack() + "[(int) " + val.temp + "]");
				}
				if(!var.referencia){
					if(!op.equals("")){
						String temp = genTemp();
						addt(temp + " = " + var.lugar + "[(int) " + var.temp + "]");
						addt(val.temp + " = " + temp + op + val.temp);
					}
					addt(var.lugar + "[(int) " + var.temp + "] = " + val.temp);
				} else {
					addt(var.temp + " = " + stack() + "[(int) " + var.temp + "]");
					if(!op.equals("")){
						String temp = genTemp();
						addt(temp + " = " + stack() + "[(int) " + var.temp + "]");
						addt(val.temp + " = " + temp + op + val.temp);
					}
					addt(stack() + "[(int) " + var.temp + "] = " + val.temp);					
				}
			} else 
				error("Se esperaba valor de tipo '" + var.tipo + "'", linea);
		} else {
			if(val.temp == null){
				String tinstancia = tnuevaInstancia(var.tipo, true, linea);
				addt(var.lugar + "[(int) " + var.temp + "] = " + tinstancia);
			} else if(var.tipo.equals(val.tipo)){
				if(val.variable)
					addt(val.temp + " = " + val.lugar + "[(int) " + val.temp + "]");
				addt(var.lugar + "[(int) " + var.temp + "] = " + val.temp);
			} else
				error("Se esperaba valor de tipo '" + var.tipo + "'", linea);
		}
	}
	
	/**
	* Realiza operación aritmética y retorna Variable con el temporal que tiene ese valor 
	* val1 valor 1
	* val2 valor 2
	* op operación aritmética a realizar
	* linea linea por posible error
	*/
	private Variable aritmética(Variable val1, Variable val2, String op, int linea) {
		Variable ret = null;
		if(val1.esNumerica() && val2.esNumerica()){
			if(val1.tipo.equals(val2.tipo)){
				String temp = val1.temp;
				if(val1.lugar == null){
					temp = genTemp();
				} else if(!val1.referencia){
					addt(val1.temp + " = " + val1.lugar + "[(int) " + val1.temp + "]");
				} else {
					addt(val1.temp + " = " + stack() + "[(int) " + val1.temp + "]");
					addt(val1.temp = " = " + stack() + "[(int) " + val1.temp + "]");
				}
				if(val2.lugar == null){
				} else if(!val2.referencia){
					addt(val2.temp + " = " + val2.lugar + "[(int) " + val2.temp + "]");
				} else {
					addt(val2.temp + " = " + stack() + "[(int) " + val2.temp + "]");
					addt(val2.temp = " = " + stack() + "[(int) " + val2.temp + "]");
				}
				addt(temp + " = " + val1.temp + op + val2.temp);
				ret = new Variable(temp, null, val1.tipo, false, false);
			} else
				error("Se debe realizar casteo", linea);
		} else 
			error("Variable no numérica, no se puede operar", linea);
		return ret;
	}
	
	/**
	* Realiza protocolo de retorno de variable. Por valor o referencia
	* var variable que tiene el temporal de valor o el temporal que apunta a lugar del valor 
	* linea por posible error
	*/
	private void retorna(Variable var, int linea){
		if((!Variable.esPrimitiva(funcion().getRetorno()) && var.tipo == null) ||
				funcion().getRetorno().equals(var.tipo)){
			String ltemp = genTemp(false);
			addt(ltemp + " = " + apstack() + " + 1");
			if(var.temp != null && var.esPrimitiva()){
				if(var.lugar == null){
				} else if(!var.referencia){
					addt(var.temp + " = " + var.lugar + "[(int) " + var.temp + "]");
				} else {
					addt(var.temp + " = " + stack() + "[(int) " + var.temp + "]");
					addt(var.temp + " = " + stack() + "[(int) " + var.temp + "]");
				}
				addt(stack() + "[(int) " + ltemp + "] = " + var.temp);
			} else {
				if(var.temp != null){
					addt(var.temp + " = " + var.lugar + "[(int) " + var.temp + "]");
					addt(stack() + "[(int) " + ltemp + "] = " + var.temp);					
				} else {
					String tinstancia = tnuevaInstancia(funcion().getRetorno(), true, linea);
					addt(stack() + "[(int) " + ltemp + "] = " + tinstancia);
				}
			}
		} else 
			error("La función debe retornar tipo '" + funcion().getRetorno() + "'", linea);
	}

	/**
	 * Genera una Variable con el resultado de la comparación booleana 
	 * de las variables.
	 * Realiza validaciones
	 * @param v1 primer valor 
	 * @param v2 segundo valor 
	 * @param op operador lógico
	 * @param linea linea por posible error
	 * @return Variable en forma de booleana resultado de aplicar la operación lógica
	 */
	private Variable lógica(Variable v1, Variable v2, String op, int linea) {
		Variable ret = null;
		if(v1.tipo.equals(v2.tipo) && v1.tipo.equals(Variable.tboolean)){
			if(op.equals(Variable.or)){
				if(v1.lugar == null){
				} else if (!v1.referencia){
					addt(v1.temp + " = " + v1.lugar + "[(int) " + v1.temp + "]");
				} else{
					addt(v1.temp + " = " + stack() + "[(int) " + v1.temp + "]");
					addt(v1.temp + " = " + stack() + "[(int) " + v1.temp + "]");
				}
				
				String temp = genTemp();
				addt(temp + " = 1");
				String ev = genEtiq();
				addtif(v1.temp, "1", ev);
				
				if(v2.lugar == null){
				} else if(!v2.referencia){
					addt(v2.temp + " = " + v2.lugar + "[(int) " + v2.temp + "]");
				} else {
					addt(v2.temp + " = " + stack() + "[(int) " + v2.temp + "]");
					addt(v2.temp + " = " + stack() + "[(int) " + v2.temp + "]");
				}
				
				addtif(v2.temp, "1", ev);
				addt(temp + " = 0");
				
				add(ev + ":\n");
				return new Variable(temp, null, Variable.tboolean, false, false);
			} else if(op.equals(Variable.and)){
				if(v1.lugar == null){
				} else if (!v1.referencia){
					addt(v1.temp + " = " + v1.lugar + "[(int) " + v1.temp + "]");
				} else {
					addt(v1.temp + " = " + stack() + "[(int) " + v1.temp + "]");
					addt(v1.temp + " = " + stack() + "[(int) " + v1.temp + "]");
				}
				
				String temp = genTemp();
				addt(temp + " = 0");
				String ev = genEtiq();
				addt("if (" + v1.temp + " == 0) goto " + ev);
				
				if(v2.lugar == null){
				} else if(!v2.referencia){
					addt(v2.temp + " = " + v2.lugar + "[(int) " + v2.temp + "]");
				} else {
					addt(v2.temp + " = " + stack() + "[(int) " + v2.temp + "]");
					addt(v2.temp + " = " + stack() + "[(int) " + v2.temp + "]");
				}
				
				addt("if (" + v2.temp + " == 0) goto " + ev);
				addt(temp + " = 1");
				
				add(ev + ":\n");
				return new Variable(temp, null, Variable.tboolean, false, false);
			} else {
				if(v1.lugar == null){
				} else if (!v1.referencia){
					addt(v1.temp + " = " + v1.lugar + "[(int) " + v1.temp + "]");
				} else {
					addt(v1.temp + " = " + stack() + "[(int) " + v1.temp + "]");
					addt(v1.temp + " = " + stack() + "[(int) " + v1.temp + "]");
				}
				
				String temp = genTemp();
				String et1 = genEtiq();
				String et2 = genEtiq();
				String ev = genEtiq();
				String ef = genEtiq();
				String es = genEtiq();
				addt("if (" + v1.temp + " == 1) goto " + et1);
				addt("goto " + et2);
				
				if(v2.lugar == null){
				} else if(!v2.referencia){
					addt(v2.temp + " = " + v2.lugar + "[(int) " + v2.temp + "]");
				} else {
					addt(v2.temp + " = " + stack() + "[(int) " + v2.temp + "]");
					addt(v2.temp + " = " + stack() + "[(int) " + v2.temp + "]");
				}
				
				add(et1 + ":\n");
				addt("if (" + v2.temp + " == 1) goto " + ef);
				addt("goto " + ev);
				add(et2 + ":\n");
				addt("if (" + v2.temp + " == 1) goto " + ev);
				addt("goto " + ef);

				add(ev + ":\n");
				addt(temp + " = 1");
				addt("goto " + es);
				add(ef + ":\n");
				addt(temp + " = 0");
				add(es + ":\n");
				
				return new Variable(temp, null, Variable.tboolean, false, false);
			}
		} else 
			error("Los operandos deben ser booleanos", linea);
		return ret;
	}

	/**
	 * Retorna una instancia tipo Variable con el temporal de la posición en 'lugar'
	 * de posición calculada de matriz
	 * @param mat variable con 'temp' definica de la forma tn-dim1[:dimm]* 
	 * @param dims string con temporales con las dimensiones separadas por ":"
	 * @param idleft linea por posible error
	 * @return Variable o null si detecta error
	 */
	private Variable variableMatriz(Variable mat, String dims, int idleft){
		Variable ret = null;
		if(mat.temp.contains(":")){
			String [] vdims = dims.split(":");
			String mattemp = mat.temp.split("-")[0];
			String [] vmax = mat.temp.split("-")[1].split(":");
			if(vmax.length == vdims.length){
				String ttemp = genTemp(false);
				addt(ttemp + " = " + vdims[0]);
				String taux = genTemp(false);
				for(int i = 1; i < vdims.length; i++){
					addt(taux + " = " + ttemp + " * " + vmax[i]);
					addt(ttemp + " = " + taux + " + " + vdims[i]);
				}
				addt(mattemp + " = " + ttemp + " + " + mattemp);
				ret = new Variable(mattemp, mat.lugar, mat.tipo, false, true);
			} else 
				error("Se requieren " + vmax.length + " dimensiones", idleft + 1);
		} else 
			error("El atributo no es matriz", idleft + 1);
		return ret;
	}

	/**
	* Realiza impresión en pantalla de dato primitivo
	* @param v2 Variable a imprimir
	*/
	private void printf(Variable v2){
		if(v2.lugar == null){
		} else if(!v2.referencia){
			addt(v2.temp + " = " + v2.lugar + "[(int) " + v2.temp + "]");
		} else {
			addt(v2.temp + " = " + stack() + "[(int) " + v2.temp + "]");
			addt(v2.temp + " = " + stack() + "[(int) " + v2.temp + "]");
		}		
		
		if(v2.tipo.equals(Variable.tchar)){
			addt("printf(\"%c\", (char)((int) " + v2.temp + "))");
		} else if(v2.tipo.equals(Variable.tint)){
			addt("printf(\"%d\", (int)" + v2.temp + ")");
		} else if(v2.tipo.equals(Variable.tfloat)){
			addt("printf(\"%lf\", " + v2.temp + ")");
		} else if(v2.tipo.equals(Variable.tboolean)){
			String lf = genEtiq();
			String ls = genEtiq();
			addtif(v2.temp, "0", lf);
			addt("printf(\"%c\", (char)((int) 116))");
			addt("printf(\"%c\", (char)((int) 114))");
			addt("printf(\"%c\", (char)((int) 117))");
			addt("printf(\"%c\", (char)((int) 101))");
			addt("goto " + ls);
			add(lf + ":\n");
			addt("printf(\"%c\", (char)((int) 102))");
			addt("printf(\"%c\", (char)((int) 97))");
			addt("printf(\"%c\", (char)((int) 108))");
			addt("printf(\"%c\", (char)((int) 115))");
			addt("printf(\"%c\", (char)((int) 101))");			
			add(ls + ":\n");
		}
		addt("printf(\"%c\", (char)((int) 10))");
	}
:};

parser code
{:
    public Symbol symbolActual() { 
        return this.cur_token;
    }

	@Override
	public void syntax_error(Symbol s) { 
		clase.error("Símbolo '" + s.value + "' fuera de contexto", archivo, s.left + 1);
//		report_error("ERR Sintaxis. Archivo: " + archivo + " Linea: " + (s.left + 1) + 
//				". Texto: \"" + s.value + "\"", null);
	}

	@Override
	public void unrecovered_syntax_error(Symbol s) throws java.lang.Exception{
		System.err.println("La Cadena: \"" + s.value + "\" en la linea: " + 
				(s.left+1) + ", columna: " + s.right + " esta fuera de contexto!!!!!" );
	}

	public ParserCup(File file, Clase clase, int pasada) throws Exception {
		super(new LexicoLex(file, clase));
		this.archivo = file.getName();
		this.clase = clase;
		this.nclase = this.archivo.split("\\.")[0];
		this.pasada = pasada;
	}
	
	protected String archivo;	
	protected Clase clase;
	protected String nclase;
	protected int pasada;
	protected Funcion funcion;
:};

terminal String PAREN, TESIS, LLA, VES, PUNTOCOMA, COR, CHETE, PUNTO, COMILLAS, 
		MAS, MENOS, MUL, DIV, MOD, ASIGNA, COMA, DOSPUNTOS, MASIGUAL, MENOSIGUAL, 
		MULIGUAL, MODIGUAL, AND, XOR, OR, COMILLA, VCHAR, VSTRING, INT, STRING, 
		IF, ELSE, ELSEIF, WHILE, DO, FOR, SWITCH, CASE, BREAK, CLASS, NEW, IMPORT, 
		PUBLIC, PROTECTED, PRIVATE, RETURN, PRINT, VFLOAT, VINT, ID, EXTENDS, VOID,
		APUNTADOR, CHAR, BOOLEAN, FLOAT, VBOOLEAN, COMPARA, PLUSPLUS, MINUSMINUS, 
		DEFAULT, NULL, THIS, NOT, NEGADO ;
		
terminal NEGATIVO ; 

non terminal S0, L, imports, _import, defclase, _extends, sentsc, sentc,
		atributo, funcion, declasig, 
		bloque, sents, sent, tupla, puntocoma, vatributo, __asigna,
		asigna, sentencias, ifelse, while, dowhile, for, _asigna,
		switchcase, tipoc, ves, bloquec ; 
non terminal Integer valorn ;
non terminal String lmatriz, if, decparam, decparams, apunt, _decparams, tipo, 
		as, case, default, cases, decl, dims ;
non terminal Character acceso ;
non terminal Variable declmatriz, variable, cvariable, valor, exp, primitivo, 
		declmatrizasigna ;
non terminal ArrayList<Variable> atrs, vatrs ;

precedence left ELSE ;
precedence left OR ;
precedence left XOR ;
precedence left AND ;
precedence left NEGADO ;
precedence left COMPARA ;
precedence left MAS, MENOS ;
precedence left MUL, DIV, MOD ;
precedence left NEGATIVO ;

start with S0 ;

S0::= L ;

L::= imports defclase 
	| defclase 
	| error {: error(); :} ;

imports::= imports _import
	| _import ;

_import::= IMPORT ID:id {:
		if(pasada() == 1) 
			clase().addClase(id, idleft + 1); 
	:} PUNTOCOMA 
	| error PUNTOCOMA {: error(); :} ;

defclase::= CLASS ID:id {: 
		if(pasada() == 1)
			clase().revisaNombre(id, idleft + 1); 
	:} _extends LLA sentsc ves:v {: if(v != null) error("Se esperaba '}'", vleft + 1); :} ;

ves::= VES | {: RESULT = "ola q ase :D"; :} ;

acceso::= PUBLIC {: if(pasada() == 1 || pasada() == 2) RESULT = Variable.vpublic; :}
	| PRIVATE {: if(pasada() == 1 || pasada() == 2) RESULT = Variable.vprivate; :}
	| PROTECTED {: if(pasada() == 1 || pasada() == 2) RESULT = Variable.vprotected; :}
	| {: if(pasada() == 1 || pasada() == 2) RESULT = Variable.vpublic; :} ;
	
_extends::= EXTENDS ID:id {: 
		if(pasada() == 1)
			clase().claseExtiende(id, idleft + 1);
	:}
	| ;

sentsc::= sentsc sentc
	| ;
	
sentc::= vatributo
	| funcion ;
	
vatributo::= atributo ASIGNA valor puntocoma:p {: if(p != null) error("Se esperaba ';'", pleft + 1); :} 
	| atributo puntocoma:p {: if(p != null) error("Se esperaba ';'", pleft + 1); :} 
	| error PUNTOCOMA {: error(); :} ;
	
atributo::= acceso:a tipo:t ID:id {:
		if(pasada() == 1)
			clase().agregaAtributo(a, t, id, idleft + 1);
	:}
	| acceso:a tipo:t lmatriz:l CHETE {:
		if(pasada() == 1 && l != null) 
			clase().agregaAtributo(a, t, l, lleft + 1);
	:} ;

puntocoma::= PUNTOCOMA | {: RESULT = "durmiendo o q ase"; :} ;
	
lmatriz::= lmatriz:l CHETE COR valorn:v {: 
		if(pasada() == 1) {
			if(v > 0){
				if (l != null)
					RESULT = l + ":" + v; 
			} else 
				error("Dimensión no válida. Se esperaba un valor positivo diferente de cero", vleft + 1);
		}
	:}
	| ID:id COR valorn:v {: 
		if(pasada() == 1) {
			if(v > 0)
				RESULT = id + "-" + v; 
			else 
				error("Dimensión no válida. Se esperaba un valor positivo diferente de cero", vleft + 1);
		}
	:} ;

valorn::= VINT:i {: if(pasada() == 1 || pasada() == 2) RESULT = Integer.valueOf(i); :} 
	| valorn:v1 MAS valorn:v2 {: if(pasada() == 1 || pasada() == 2) RESULT = v1 + v2; :}
	| valorn:v1 MENOS valorn:v2 {: if(pasada() == 1 || pasada() == 2) RESULT = v1 - v2; :}
	| valorn:v1 MUL valorn:v2 {: if(pasada() == 1 || pasada() == 2) RESULT = v1 * v2; :}
	| valorn:v1 DIV valorn:v2 {: if(pasada() == 1 || pasada() == 2) RESULT = v1 / v2; :}
	| valorn:v1 MOD valorn:v2 {: if(pasada() == 1 || pasada() == 2) RESULT = v1 % v2; :}
	| PAREN valorn:v TESIS {: if(pasada() == 1 || pasada() == 2) RESULT = v; :}
	| MENOS valorn:v {: if(pasada() == 1 || pasada() == 2) RESULT = - v; :} %prec NEGATIVO 
	| error {: RESULT = 0; :} ;

tipo::= INT {: if(pasada() == 1 || pasada() == 2) RESULT = Variable.tint; :}
	| STRING {: if(pasada() == 1 || pasada() == 2) RESULT = Variable.tString; :}
	| CHAR {: if(pasada() == 1 || pasada() == 2) RESULT = Variable.tchar; :}
	| BOOLEAN {: if(pasada() == 1 || pasada() == 2) RESULT = Variable.tboolean; :}
	| FLOAT {: if(pasada() == 1 || pasada() == 2) RESULT = Variable.tfloat; :}
	| ID:id {: if(pasada() == 1 || pasada() == 2) RESULT = id; :} ;

tipoc::= INT {: if(pasada() == 1 || pasada() == 2) RESULT = Variable.tint; :}
	| STRING {: if(pasada() == 1 || pasada() == 2) RESULT = Variable.tString; :}
	| CHAR {: if(pasada() == 1 || pasada() == 2) RESULT = Variable.tchar; :}
	| BOOLEAN {: if(pasada() == 1 || pasada() == 2) RESULT = Variable.tboolean; :}
	| FLOAT {: if(pasada() == 1 || pasada() == 2) RESULT = Variable.tfloat; :} ;
	
funcion::= acceso:a tipo:t ID:i PAREN {:
		if(pasada() == 2) {
			funcion(new Funcion(clase(), t));
			add("\nvoid " + nclase() + "_" + t + "_" + i);
		}
	:} _decparams:p {:
		if(pasada() == 1)
			if(Variable.esPrimitiva(t))
				clase().addFuncion(a, t, i, p, ileft + 1);
			else
				error("Tipo de retorno inválido para la función", tleft + 1);
	:} TESIS bloquec {: if(pasada() == 2) { add("}\n"); resetcTemp(); } :}
	| acceso:a VOID:t ID:i PAREN {:
		if(pasada() == 2) {
			funcion(new Funcion(clase(), t));
			add("\nvoid " + nclase() + "_" + t + "_" + i);
		}
	:} _decparams:p {:
		if(pasada() == 1)
			clase().addFuncion(a, t, i, p, ileft + 1);
	:} TESIS bloquec {: if(pasada() == 2) { add("}\n"); resetcTemp(); } :} 
	| acceso:a ID:i PAREN {:
		if(pasada() == 2) {
			funcion(new Funcion(clase(), Funcion.rconstructor));
			add("\nvoid " + nclase() + "_void_" + i);
		}
	:} _decparams:p {:
		if(pasada() == 1)
			clase().addFuncion(a, "%constructor%", i, p, ileft + 1);
	:} TESIS bloquec {: if(pasada() == 2) { add("}\n"); resetcTemp(); } :} 
	| error {: error(); :} ;

_decparams::= decparams:d {: 
		if (pasada() == 1) 
			RESULT = d; 
		else if (pasada() == 2) 
			add("() {\n");
	:}
	| {: 
		if(pasada() == 1) 
			RESULT = ""; 
		else if (pasada() == 2)
			add("() {\n");
	:} ;

decparams::= decparams:ds COMA decparam:d {: RESULT = ds + ":" + d; :}
	| decparam:d {: RESULT = d; :} ;
	
decparam::= tipo:t apunt:a ID:id {: 
		if(pasada() == 1) 
			RESULT = a + t; 
		else if(pasada() == 2) {
			funcion().addVariable(t, id, idleft + 1);
			add("_" + t);
		}
	:} ;

apunt::= APUNTADOR {: if(pasada() == 1) RESULT = "&"; :}
	| {: if(pasada() == 1) RESULT = " "; :} ;

bloquec::= LLA {:
		if(pasada() == 2)
			etSalidaMetodo = genEtiq();
	:} sents VES {:
		if(pasada() == 2){
			add(etSalidaMetodo + ":\n");
			addt("return");
		}
	:} ;

bloque::= LLA {:
		if(pasada() == 2)
			funcion(funcion().agregarAmbito());
	:} sents VES {:
		if(pasada() == 2)
			funcion(funcion().ambitoSuperior());
	:} ;

sentencias::= sent:s {:
		if(pasada() == 2 && s != null) 
			error("No se pueden hacer declaraciones", sleft + 1);
	:}
	| bloque ;
	
sents::= sents sent {: if(pasada() == 2) resetTempPorGuardar(); :}
	| ;

declasig::= decl:svar ASIGNA valor:val {:
		if(pasada() == 2) 
			if(svar != null && val != null)
				asigna(buscaEnAmbito(svar), val, "", svarleft + 1);
	:}
	| decl 
	| declmatrizasigna VES
	| declmatriz:d CHETE {:
		if(pasada() == 2)
			if(d != null)
				funcion().addVariable(d.tipo, d.temp, dleft + 1);
	:} ;
	
declmatrizasigna::= declmatrizasigna COMA valor 
	| declmatriz:dec CHETE ASIGNA LLA valor:val {:
		if(pasada() == 2){
			if(dec != null){
				funcion().addVariable(dec.tipo, dec.temp, decleft + 1);
				
				if(val != null){ 
					
				}
			}
		}
	:} ;

decl::= tipo:t ID:id {:
		if(pasada() == 2){
			if(funcion().addVariable(t, id, idleft + 1)){
				RESULT = id;
			}
		}
	:} ;

declmatriz::= declmatriz:d CHETE COR valorn:v {:
		if(pasada() == 2)
			if(d != null){
				d.temp += ":" + v;
				RESULT = d;
			}
	:}
	| tipo:t ID:id COR valorn:v {:
		if(pasada() == 2)
			if(v > 0){
				RESULT = new Variable(id + "-" + v, null, t, 
						false, false);
			} else
				error("Se esperaba dimensión de matriz positiva", idleft + 1);
			
	:} ;

as::= ASIGNA {: if(pasada() == 2) RESULT = ""; :}
	| MASIGUAL {: if(pasada() == 2) RESULT = " + "; :}
	| MENOSIGUAL {: if(pasada() == 2) RESULT = " - "; :}
	| MULIGUAL {: if(pasada() == 2) RESULT = " * "; :}
	| MODIGUAL {: if(pasada() == 2) RESULT = " % "; :} ;

sent::= declasig PUNTOCOMA {: RESULT = "42 >:)"; :}
	| asigna PUNTOCOMA
	| variable PUNTOCOMA
	| RETURN valor:v PUNTOCOMA {:
		if(pasada() == 2){
			if(v != null)
				retorna(v, vleft + 1);
		}
	:}
	| ifelse
	| while
	| dowhile PUNTOCOMA
	| switchcase 
	| BREAK:v PUNTOCOMA {:
		if(pasada() == 2)
			if(idisplay >= 0){
				addt("goto " + display[idisplay]);
			} else 
				error("Sentencia BREAK en ámbito incorrecto", vleft + 1);
	:}
	| for 
	| PRINT PAREN valor:v TESIS PUNTOCOMA {:
		if(pasada() == 2)
			if(v != null)
				if(v.temp != null){
					if(v.esPrimitiva()){
						printf(v);
					} else if(v.tipo.equals(Variable.tString)){
						String temp = genTemp(false);
						addt(temp + " = " + v.lugar + "[(int) " + v.temp + "]");
						String li = genEtiq();
						String ls = genEtiq();
						add(li + ":\n");
						addt(temp + " = " + temp + " + 1");
						addt(v.temp + " = " + heap() + "[(int) " + temp + "]");
						addtif(v.temp, "0", ls);
						addt("printf(\"%c\", (char)((int) " + v.temp + "))");
						addt("goto " + li);
						add(ls + ":\n");
						addt("printf(\"%c\", (char)((int) 10))");
					}
				} else
					error("Se variable no null", vleft + 1);
	:}
	| error PUNTOCOMA {: error(); :} ; 

asigna::= variable:var as:as valor:val {:
		if(pasada() == 2)
			if(var != null && val != null)
				if(var.variable)
					asigna(var, val, as, varleft + 1);
				else 
					error("Se esperaba variable para hacer asignación", varleft + 1);
	:}
	| cvariable ;
	
variable::= NEW ID:id PAREN vatrs:v TESIS {:
		if(pasada() == 2){
			String temp = tnuevaInstancia(id, idleft + 1);
			if(temp != null){
				RESULT = new Variable(temp, heap(), id, true, false);
				llamarFuncion(id, id, v, temp, idleft + 1);
			}
		}
	:}
	| THIS PUNTO ID:id COR dims:dims CHETE {:
		if(pasada() == 2 && dims != null){
			String temp = genTemp();
			addt(temp + " = " + stack() + "[(int) " + apstack() + "]");
			Variable mat = buscaEnInstancia(id, nclase(), temp, idleft + 1);
			if(mat != null)
				RESULT = variableMatriz(mat, dims, idleft + 1);
		}
	:}
	| ID:id COR dims:dims CHETE {:
		if(pasada() == 2 && dims != null){
			Variable mat = buscaEnAmbito(id);
			if(mat == null){
				String temp = genTemp();
				addt(temp + " = " + stack() + "[(int) " + apstack() + "]");
				mat = buscaEnInstancia(id, nclase(), temp, idleft + 1);
				if(mat != null)
					RESULT = variableMatriz(mat, dims, idleft + 1);				
			} else
				RESULT = variableMatriz(mat, dims, idleft + 1);
		}
	:}
	| variable:var PUNTO ID:id COR dims:dims CHETE {:
		if(pasada() == 2 && dims != null && var != null){
			
		}
	:}
	| THIS PUNTO ID:id {:
		if(pasada() == 2){
			String temp = genTemp();
			addt(temp + " = " + stack() + "[(int) " + apstack() + "]");
			RESULT = buscaEnInstancia(id, nclase(), temp, idleft + 1);
			if(RESULT != null && RESULT.temp.contains(":")){
				error("Acceso ilegal a la matriz '" + id + "'", idleft + 1);
				RESULT = null;
			}
		}
	:}
	| ID:id {:
		if(pasada() == 2){
			RESULT = buscaEnAmbito(id);
			if(RESULT == null){
				String temp = genTemp();
				addt(temp + " = " + stack() + "[(int) " + apstack() + "]");
				RESULT = buscaEnInstancia(id, nclase(), temp, idleft + 1);
			}
			if(RESULT != null && RESULT.temp.contains(":")){
				error("Acceso ilegal a la matriz '" + id + "'", idleft + 1);
				RESULT = null;
			}
		}
	:}
	| variable:v PUNTO ID:id {:
		if(pasada() == 2){
			if(v != null){
				addt(v.temp + " = " + heap() + "[(int) " + v.temp + "]");
				RESULT = buscaEnInstancia(id, v.tipo, v.temp, idleft + 1);
				if(RESULT != null && RESULT.temp.contains(":")){
					error("Acceso ilegal a la matriz '" + id + "'", idleft + 1);
					RESULT = null;
				}
			}
		}
	:}
	| THIS PUNTO ID:id PAREN vatrs:v TESIS {:
		if(pasada() == 2){
			String temp = genTemp();
			addt(temp + " = " + stack() + "[(int) " + apstack() + "]");
			RESULT = llamarFuncion(nclase(), id, v, temp, idleft + 1);
		}
	:}
	| ID:id PAREN vatrs:v TESIS {:
		if(pasada() == 2){
			String temp = genTemp();
			addt(temp + " = " + stack() + "[(int) " + apstack() + "]");
			RESULT = llamarFuncion(nclase(), id, v, temp, idleft + 1);
		}
	:}
	| variable:var PUNTO ID:id PAREN vatrs:v TESIS {:
		if(pasada() == 2){
			if(v != null && var != null){
				RESULT = llamarFuncion(var.tipo, id, v, var.temp, idleft + 1);
			}
		}
	:} ;

dims::= dims:d CHETE COR valor:v {:
		if(pasada() == 2)
			if(v != null && d != null)
				if(v.temp != null && v.tipo.equals(Variable.tint)){
					if(v.lugar == null){
					} else if(!v.referencia){
						addt(v.temp + " = " + v.lugar + "[(int) " + v.temp + "]");
					} else {
						addt(v.temp + " = " + stack() + "[(int) " + v.temp + "]");
						addt(v.temp + " = " + stack() + "[(int) " + v.temp + "]");
					}
					RESULT = d + ":" + v.temp;
				} else 
					error("Se espera expresión tipo INT", vleft + 1);
	:} 
	| valor:v {:
		if(pasada() == 2)
			if(v != null)
				if(v.temp != null && v.tipo.equals(Variable.tint)){
					if(v.lugar == null){
					} else if(!v.referencia){
						addt(v.temp + " = " + v.lugar + "[(int) " + v.temp + "]");
					} else {
						addt(v.temp + " = " + stack() + "[(int) " + v.temp + "]");
						addt(v.temp + " = " + stack() + "[(int) " + v.temp + "]");
					}
					RESULT = v.temp;
				} else 
					error("Se espera expresión tipo INT", vleft + 1);
	:} ;

cvariable::= variable:v PLUSPLUS {:
		if(pasada() == 2)
			if(v != null)
				if(v.variable && (v.tipo.equals(Variable.tfloat) || v.tipo.equals(Variable.tint))){
					RESULT = cvariable(false, " + ", v, vleft + 1);
				} else
					error("Se esperaba variable numérica", vleft + 1);
	:}
	| variable:v MINUSMINUS {:
		if(pasada() == 2)
			if(v != null)
				if(v.variable && (v.tipo.equals(Variable.tfloat) || v.tipo.equals(Variable.tint)))
					RESULT = cvariable(false, " - ", v, vleft + 1);
				else
					error("Se esperaba variable numérica", vleft + 1);
	:}
	| PLUSPLUS variable:v {:
		if(pasada() == 2)
			if(v != null)
				if(v.variable && (v.tipo.equals(Variable.tfloat) || v.tipo.equals(Variable.tint)))
					RESULT = cvariable(true, " + ", v, vleft + 1);
				else
					error("Se esperaba variable numérica", vleft + 1);
	:}
	| MINUSMINUS variable:v {:
		if(pasada() == 2)
			if(v != null)
				if(v.variable && (v.tipo.equals(Variable.tfloat) || v.tipo.equals(Variable.tint)))
					RESULT = cvariable(true, " - ", v, vleft + 1);
				else
					error("Se esperaba variable numérica", vleft + 1);
	:} ;
	
vatrs::= atrs:l {: if(pasada() == 2) RESULT = l; :}
	| {: if(pasada() == 2) RESULT = new ArrayList<Variable>(); :} ;

atrs::= atrs:l COMA valor:v {:
		if(pasada() == 2){
			RESULT = l;
			if(v != null)
				RESULT.add(v);
		}
	:}
	| valor:v {: 
		if(pasada() == 2) {
			RESULT = new ArrayList<Variable>();
			if(v != null)
				RESULT.add(v);
		}
	:} ;

valor::= valor:v1 MAS valor:v2 {:
		if(pasada() == 2)
			if(v1 != null && v2 != null)
				RESULT = aritmética(v1, v2, " + ", v1left + 1);
	:}
	| valor:v1 MENOS valor:v2 {:
		if(pasada() == 2)
			if(v1 != null && v2 != null)
				RESULT = aritmética(v1, v2, " - ", v1left + 1);
	:}
	| valor:v1 MUL valor:v2 {:
		if(pasada() == 2)
			if(v1 != null && v2 != null)
				RESULT = aritmética(v1, v2, " * ", v1left + 1);
	:}
	| valor:v1 DIV valor:v2{:
		if(pasada() == 2)
			if(v1 != null && v2 != null)
				RESULT = aritmética(v1, v2, " / ", v1left + 1);
	:}
	| valor:v1 MOD valor:v2 {:
		if(pasada() == 2)
			if(v1 != null && v2 != null)
				RESULT = aritmética(v1, v2, " % ", v1left + 1);
	:}
	| MENOS valor:v {:
		if(pasada() == 2)
			if(v != null)
				RESULT = aritmética(new Variable("0", null, v.tipo, false, false), 
						v, " - ", vleft + 1);
	:} %prec NEGATIVO 
	| valor:v1 COMPARA:com valor:v2 {:
		if(pasada() == 2)
			if(v1 != null && v2 != null)
				if(v1.esNumerica() && v2.esNumerica()){
					if(v1.lugar == null){
					} else if(!v1.referencia){
						addt(v1.temp + " = " + v1.lugar + "[(int) " + v1.temp + "]");
					} else {
						addt(v1.temp + " = " + stack() + "[(int) " + v1.temp + "]");
						addt(v1.temp + " = " + stack() + "[(int) " + v1.temp + "]");
					}
					
					if(v2.lugar == null){
					} else if(!v2.referencia){
						addt(v2.temp + " = " + v2.lugar + "[(int) " + v2.temp + "]");
					} else {
						addt(v2.temp + " = " + stack() + "[(int) " + v2.temp + "]");
						addt(v2.temp + " = " + stack() + "[(int) " + v2.temp + "]");
					}
					
					String temp = genTemp();
					String et = genEtiq();
					addt(temp + " = 1");
					com = com.equals("#") ? "==" : com;
					addt("if (" + v1.temp + " " + com + " " + v2.temp +") goto " + et);
					addt(temp + " = 0");
					add(et + ":\n");
					
					RESULT = new Variable(temp, null, Variable.tboolean, false, false);
				} else
					error("Los valores deben ser numéricos", v1left + 1);
	:}
	| valor:v1 OR valor:v2 {:
		if(pasada() == 2)
			if(v1 != null && v2 != null)
				if(v1.temp != null && v2.temp != null)
					RESULT = lógica(v1, v2, Variable.or, v1left + 1);
				else
					error("No se permiten valores nulos", v1left + 1);
	:}
	| valor:v1 AND valor:v2 {:
		if(pasada() == 2)
			if(v1 != null && v2 != null)
				if(v1.temp != null && v2.temp != null)
					RESULT = lógica(v1, v2, Variable.and, v1left + 1);
				else
					error("No se permiten valores nulos", v1left + 1);	
	:}
	| valor:v1 XOR valor:v2 {:
		if(pasada() == 2)
			if(v1 != null && v2 != null)
				if(v1.temp != null && v2.temp != null)
					RESULT = lógica(v1, v2, Variable.xor, v1left + 1);
				else
					error("No se permiten valores nulos", v1left + 1);
	:}
	| NOT valor:v {:
		if(pasada() == 2)
			if(v != null)
				if(v.temp != null && v.tipo.equals(Variable.tboolean)){
					if(v.lugar == null){
					} else if (!v.referencia){
						addt(v.temp + " = " + v.lugar + "[(int) " + v.temp + "]");
					} else{
						addt(v.temp + " = " + stack() + "[(int) " + v.temp + "]");
						addt(v.temp + " = " + stack() + "[(int) " + v.temp + "]");
					}
					String temp = genTemp();
					String et = genEtiq();
					addt(temp + " = 0");
					addtif(v.temp, "1", et);
					addt(temp + " = 1");
					add(et + ":\n");
					RESULT = new Variable(temp, null, Variable.tboolean, false, false);
				} else
					error("Variable debe ser booleana", vleft + 1);
	:} %prec NEGADO 
	| PAREN valor:v TESIS {: if(pasada() == 2) RESULT = v; :}
	| PAREN tipoc:t TESIS valor:v {:
		if(pasada() == 2)
			if(v != null)
				if(v.temp != null && v.esPrimitiva()){
					if(t.equals(Variable.tboolean)){
						String lf = genEtiq();
						addt(v.temp + " = 0");
						addtif(v.temp, "0", lf);
						addt(v.temp + " = 1");
						add(lf + ":\n");
						v.tipo = Variable.tboolean;
						RESULT = v;
					} else if(t.equals(Variable.tint)){
						addt(v.temp + " = (int) " + v.temp);
						v.tipo = Variable.tint;
						RESULT = v;
					} else if(t.equals(Variable.tfloat)){
						v.tipo = Variable.tfloat;
						RESULT = v;
					} else if(t.equals(Variable.tchar)){
						addt(v.temp + " = (int) " + v.temp);
						RESULT = v;
					}
				} else
					error("Se esperaba un valor de tipo primitivo", vleft + 1);
	:}
	| exp:e {: if(pasada() == 2) RESULT = e; :}
	| error {: error(); :} ;
	
exp::= primitivo:v {: if(pasada() == 2) RESULT = v; :}
	| NULL {: if(pasada() == 2) RESULT = new Variable(null, null, null, false, false); :}
	| variable:v {: if(pasada() == 2) RESULT = v; :}
	| cvariable:v {: if(pasada() == 2) RESULT = v; :} ;
	
primitivo::= COMILLAS VSTRING:v COMILLAS {:
		if(pasada() == 2) {
			String temp = tnuevaInstanciaString(v, vleft + 1);
			RESULT = new Variable(temp, heap(), Variable.tString, true, false);
		}
	:}
	| COMILLA VCHAR:v COMILLA {:
		if(pasada() == 2)
			RESULT = new Variable(String.valueOf((int) v.charAt(0)), null, Variable.tchar, false, false);
	:}
	| VINT:v {:
		if(pasada() == 2)
			RESULT = new Variable(v, null, Variable.tint, false, false);
	:}
	| VFLOAT:v {:
		if(pasada() == 2)
			RESULT = new Variable(v, null, Variable.tfloat, false, false);
	:}
	| VBOOLEAN:v {:
		if(pasada() == 2)
			RESULT = new Variable(v.equals("true") ? "1" : "0", null, Variable.tboolean, false, false);
	:};
	
ifelse::= if:i ELSE {:
		if(pasada() == 2)
			if(i != null){
				String ls = genEtiq();
				addt("goto " + ls);
				push(ls);
				add(i + ":\n");
			}
	:} sentencias {:
		if(pasada() == 2)
			if(i != null){
				String ls = pop();
				add(ls + ":\n");
			}
	:}
	| if:i {:
		if(pasada() == 2)
			if(i != null)
				add(i + ":\n");
	:} ;

if::= IF PAREN valor:v TESIS {:
		if(pasada() == 2)
			if(v != null){
				if(v.temp != null && v.tipo.equals(Variable.tboolean)){
					if(v.lugar == null){
					} else if(!v.referencia){
						addt(v.temp + " = " + v.lugar + "[(int) " + v.temp + "]");
					} else {
						addt(v.temp + " = " + stack() + "[(int) " + v.temp + "]");
						addt(v.temp + " = " + stack() + "[(int) " + v.temp + "]");
					}
					String lf = genEtiq();
					addtif(v.temp, "0", lf);
					RESULT = lf;
				} else
					error("Se espera expresión booleana", vleft + 1);
			}
	:} sentencias ;
	
while::= WHILE PAREN {:
		if(pasada() == 2){
			idisplay ++;
			String li = genEtiq();
			push(li);
			add(li + ":\n");
		}
	:} valor:v {:
		if(pasada() == 2)
			if(v != null)
				if(v.temp != null && v.tipo.equals(Variable.tboolean)){
					if(v.lugar == null){
					} else if(!v.referencia){
						addt(v.temp + " = " + v.lugar + "[(int) " + v.temp + "]");
					} else {
						addt(v.temp + " = " + stack() + "[(int) " + v.temp + "]");
						addt(v.temp + " = " + stack() + "[(int) " + v.temp + "]");
					}
					String ls = genEtiq();
					display[idisplay] = ls;
					addtif(v.temp, "0", ls);
				} else
					error("Se espera expresión booleana", vleft + 1);
	:} TESIS sentencias {: 
		if(pasada() == 2){
			String li = pop();
			addt("goto " + li);
			add(display[idisplay] + ":\n");
			idisplay --; 
		}
	:} ;

dowhile::= DO {:
		if(pasada() == 2){
			String li = genEtiq();
			push(li);
			add(li + ":\n");
			String ls = genEtiq();
			display[++ idisplay] = ls;
		}
	:} sentencias WHILE PAREN valor:v {:
		if(pasada() == 2){
			if(v != null){
				if(v.temp != null & v.tipo.equals(Variable.tboolean)){
					if(v.lugar == null){
					} else if(!v.referencia){
						addt(v.temp + " = " + v.lugar + "[(int) " + v.temp + "]");
					} else {
						addt(v.temp + " = " + stack() + "[(int) " + v.temp + "]");
						addt(v.temp + " = " + stack() + "[(int) " + v.temp + "]");
					}
					String li = pop();
					String ls = display[idisplay --];
					addtif(v.temp, "1", li);
					add(ls + ":\n");
				} else
					error("Se espera expresión booleana", vleft + 1);
			}
		}
	:} TESIS ;

for::= FOR PAREN {:
		if(pasada() == 2)
			funcion(funcion().agregarAmbito());
	:} __asigna PUNTOCOMA {:
		if(pasada() == 2){
			String li = genEtiq();
			push(li);
			add(li + ":\n");
		}
	:} valor:v {:
		if(pasada() == 2)
			if(v != null)
				if(v.temp != null && v.tipo.equals(Variable.tboolean)){
					if(v.lugar == null){
					} else if (!v.referencia){
						addt(v.temp + " = " + v.lugar + "[(int) " + v.temp + "]");
					} else{
						addt(v.temp + " = " + stack() + "[(int) " + v.temp + "]");
						addt(v.temp + " = " + stack() + "[(int) " + v.temp + "]");
					}
					
					String ls = genEtiq();
					display[++ idisplay] = ls;
					addtif(v.temp, "0", ls);
				} else
					error("Se esperaba expresión booleana", vleft + 1);
	:} PUNTOCOMA _asigna TESIS sentencias {:
		if(pasada() == 2){
			String li = pop();
			addt("goto " + li);
			funcion(funcion().ambitoSuperior());
			add(display[idisplay --] + ":\n");
		}
	:};

__asigna::= _asigna
	| declasig ;

_asigna::= asigna 
	| ;

switchcase::= SWITCH PAREN valor:v {:
		if(pasada() == 2)
			if(v != null)
				if(v.variable && (v.tipo.equals(Variable.tboolean)
						|| v.tipo.equals(Variable.tfloat) || v.tipo.equals(Variable.tint))){
					if(v.lugar == null){
					} else if(!v.referencia){
						addt(v.temp + " = " + v.lugar + "[(int) " + v.temp + "]");
					} else {
						addt(v.temp + " = " + stack() + "[(int) " + v.temp + "]");
						addt(v.temp + " = " + stack() + "[(int) " + v.temp + "]");
					}
					String et = genEtiq();
					push(et);
					addt("goto " + et);
					String ls = genEtiq();
					display[++ idisplay] = ls;
					cases[++ icases] = v;
				} else
					error("Tipo inválido en SWITCH CASE");
	:} TESIS LLA cases:c default:d {: 
		if(pasada() == 2 && c != null && d != null){
			add(pop() + ":\n");
			add(c + d);
			add(display[idisplay --] + ":\n");
			icases --;
		}
	:} VES ;

cases::= cases:cs case:c {: if(pasada() == 2) if(cs != null && c != null) RESULT = cs + c; :}
	| case:c {: if(pasada() == 2) RESULT = c; :} ;
	
case::= CASE {:
		if(pasada() == 2)
			funcion(funcion().agregarAmbito());
	:} valor:v DOSPUNTOS {:
		if(v != null)
			if(v.lugar == null){
				Variable sw = cases[icases];
				if(sw.tipo.equals(v.tipo)){
					String et = genEtiq();
					RESULT = "\tif(" + sw.temp + " == " + v.temp + ") goto " + et + "\n";
					add(et + ":\n");
				} else
					error("Tipo colocado en caso no es válido", vleft + 1);
			} else 
				error("El valor debe ser constante", vleft + 1);
	:} sents {:
		if(pasada() == 2)
			funcion(funcion().ambitoSuperior());
	:};

default::= DEFAULT DOSPUNTOS {:
		if(pasada() == 2){
			String et = genEtiq();
			RESULT = "\tgoto " + et + "\n";
			add(et + ":\n");
		}
	:} sents 
	| {: if(pasada() == 2) RESULT = ""; :};





//// arreglar si manda null como atributo de clase 
///
